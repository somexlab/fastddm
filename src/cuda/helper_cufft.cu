// Maintainer: enrico-lattuada

/*! \file helper_cufft.cu
    \brief Definition of cufft helper functions
*/

// *** headers ***
#include "helper_cufft.cuh"
#include "helper_debug.cuh"

// *** code ***

/*
    Create the cufft plan for the fft2.
 */
hipfftHandle fft2_create_plan(size_t nx,
                             size_t ny,
                             size_t batch,
                             size_t pitch,
                             bool is_double_prec)
{
    // Define parameters
    int rank = 2;                                            // The rank of the fft (2 = fft2)
    int n[2] = {(int)ny, (int)nx};                           // Dimensions
    int inembed[2] = {(int)ny, 2 * (int)pitch};              // nembed input values (NULL is equivalent to passing n)
    int istride = 1;                                         // Distance between two elements in the input
    int idist = (int)(2 * ny * pitch);                       // Distance between k-th and (k+1)-th input elements
    int onembed[2] = {(int)ny, (int)pitch};                  // nembed output values (NULL is equivalent to passing n)
    int ostride = 1;                                         // Distance between two elements in the output
    int odist = (int)(ny * pitch);                           // Distance between k-th and (k+1)-th output elements
    hipfftType type = is_double_prec ? HIPFFT_D2Z : HIPFFT_R2C; // Fft type (real to complex)

    // Create the fft2 plan
    hipfftHandle plan;
    cufftSafeCall(hipfftPlanMany(&plan,
                                rank,
                                n,
                                inembed,
                                istride,
                                idist,
                                onembed,
                                ostride,
                                odist,
                                type,
                                (int)batch));

    return plan;
}

/*!
    Evaluate the device memory size in bytes for fft2
 */
void fft2_get_mem_size(size_t nx,
                       size_t ny,
                       size_t batch,
                       size_t pitch,
                       bool is_double_prec,
                       size_t *memsize,
                       hipfftResult &cufft_res)
{
    // Define parameters
    int rank = 2;                                            // The rank of the fft (2 = fft2)
    int n[2] = {(int)ny, (int)nx};                           // Dimensions
    int inembed[2] = {(int)ny, 2 * (int)pitch};              // nembed input values (NULL is equivalent to passing n)
    int istride = 1;                                         // Distance between two elements in the input
    int idist = (int)(2 * ny * pitch);                       // Distance between k-th and (k+1)-th input elements
    int onembed[2] = {(int)ny, (int)pitch};                  // nembed output values (NULL is equivalent to passing n)
    int ostride = 1;                                         // Distance between two elements in the output
    int odist = (int)(ny * pitch);                           // Distance between k-th and (k+1)-th output elements
    hipfftType type = is_double_prec ? HIPFFT_D2Z : HIPFFT_R2C; // Fft type (real to complex)

    // Create the fft2 plan
    hipfftHandle plan;
    cufftSilentSafeCall(hipfftPlanMany(&plan,
                                      rank,
                                      n,
                                      inembed,
                                      istride,
                                      idist,
                                      onembed,
                                      ostride,
                                      odist,
                                      type,
                                      (int)batch));

    // Evaluate the memory size
    cufft_res = hipfftGetSizeMany(plan,
                                 rank,
                                 n,
                                 inembed,
                                 istride,
                                 idist,
                                 onembed,
                                 ostride,
                                 odist,
                                 type,
                                 (int)batch,
                                 memsize);

    cufftSilentSafeCall(hipfftDestroy(plan));
}

/*
    Create the cufft plan for the complex to complex fft.
 */
hipfftHandle fft_create_plan(size_t nt,
                            size_t batch,
                            size_t pitch,
                            bool is_double_prec)
{
    // Define parameters
    int rank = 1;                                            // The rank of the fft (1 = fft)
    int n[1] = {(int)nt};                                    // Dimensions
    int inembed[] = {(int)pitch};                            // NULL is equivalent to passing n
    int istride = 1;                                         // Distance between two elements in the input
    int idist = (int)pitch;                                  // Distance between k-th and (k+1)-th input elements
    int onembed[] = {(int)pitch};                            // NULL is equivalent to passing n
    int ostride = 1;                                         // Distance between two elements in the output
    int odist = (int)pitch;                                  // Distance between k-th and (k+1)-th output elements
    hipfftType type = is_double_prec ? HIPFFT_Z2Z : HIPFFT_C2C; // Fft type (complex to complex)

    // Create the fft plan
    hipfftHandle plan;
    cufftSafeCall(hipfftPlanMany(&plan,
                                rank,
                                n,
                                inembed,
                                istride,
                                idist,
                                onembed,
                                ostride,
                                odist,
                                type,
                                (int)batch));

    return plan;
}

/*!
    Evaluate the device memory size in bytes for fft
 */
void fft_get_mem_size(size_t nt,
                      size_t batch,
                      size_t pitch,
                      bool is_double_prec,
                      size_t *memsize,
                      hipfftResult &cufft_res)
{
    // Define parameters
    int rank = 1;                                            // The rank of the fft (1 = fft)
    int n[1] = {(int)nt};                                    // Dimensions
    int *inembed = NULL;                                     // NULL is equivalent to passing n
    int istride = 1;                                         // Distance between two elements in the input
    int idist = (int)pitch;                                  // Distance between k-th and (k+1)-th input elements
    int *onembed = NULL;                                     // NULL is equivalent to passing n
    int ostride = 1;                                         // Distance between two elements in the output
    int odist = (int)pitch;                                  // Distance between k-th and (k+1)-th output elements
    hipfftType type = is_double_prec ? HIPFFT_Z2Z : HIPFFT_C2C; // Fft type (complex to complex)

    // Create the fft plan
    hipfftHandle plan;
    cufftSilentSafeCall(hipfftPlanMany(&plan,
                                      rank,
                                      n,
                                      inembed,
                                      istride,
                                      idist,
                                      onembed,
                                      ostride,
                                      odist,
                                      type,
                                      (int)batch));

    // Evaluate the memory size
    cufft_res = hipfftGetSizeMany(plan,
                                 rank,
                                 n,
                                 inembed,
                                 istride,
                                 idist,
                                 onembed,
                                 ostride,
                                 odist,
                                 type,
                                 (int)batch,
                                 memsize);

    cufftSilentSafeCall(hipfftDestroy(plan));
}
