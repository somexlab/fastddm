#include "hip/hip_runtime.h"
// Maintainer: enrico-lattuada

/*! \file helper_ddm_cuda.cu
    \brief Definition of helper functions for Differential Dynamic Microscopy on the GPU
*/

// *** headers ***
#include "helper_ddm_cuda.cuh"
#include "helper_debug.cuh"

#include <hip/hip_runtime.h>

#include <hip/hip_cooperative_groups.h>
namespace cg = cooperative_groups;

// *** code ***
const unsigned long long TILE_DIM = 32;  // leave this unchanged!
const unsigned long long BLOCK_ROWS = 8; // leave this unchanged!

/*!
    Convert array from T to double on device and prepare for fft2
*/
template <typename T>
__global__ void copy_convert_kernel(T *d_in,
                                    double *d_out,
                                    unsigned long long width,
                                    unsigned long long height,
                                    unsigned long long length,
                                    unsigned long long ipitch,
                                    unsigned long long idist,
                                    unsigned long long opitch,
                                    unsigned long long odist)
{
    for (unsigned long long t = blockIdx.x; t < length; t += gridDim.x)
    {
        for (unsigned long long y = blockIdx.y; y < height; y += gridDim.y)
        {
            for (unsigned long long x = threadIdx.x; x < width; x += blockDim.x)
            {
                d_out[t * odist + y * opitch + x] = (double)d_in[t * idist + y * ipitch + x];
            }
        }
    }
}

template __global__ void copy_convert_kernel<u_int8_t>(u_int8_t *d_in, double *d_out, unsigned long long width, unsigned long long height, unsigned long long length, unsigned long long ipitch, unsigned long long idist, unsigned long long opitch, unsigned long long odist);
template __global__ void copy_convert_kernel<int16_t>(int16_t *d_in, double *d_out, unsigned long long width, unsigned long long height, unsigned long long length, unsigned long long ipitch, unsigned long long idist, unsigned long long opitch, unsigned long long odist);
template __global__ void copy_convert_kernel<u_int16_t>(u_int16_t *d_in, double *d_out, unsigned long long width, unsigned long long height, unsigned long long length, unsigned long long ipitch, unsigned long long idist, unsigned long long opitch, unsigned long long odist);
template __global__ void copy_convert_kernel<int32_t>(int32_t *d_in, double *d_out, unsigned long long width, unsigned long long height, unsigned long long length, unsigned long long ipitch, unsigned long long idist, unsigned long long opitch, unsigned long long odist);
template __global__ void copy_convert_kernel<u_int32_t>(u_int32_t *d_in, double *d_out, unsigned long long width, unsigned long long height, unsigned long long length, unsigned long long ipitch, unsigned long long idist, unsigned long long opitch, unsigned long long odist);
template __global__ void copy_convert_kernel<int64_t>(int64_t *d_in, double *d_out, unsigned long long width, unsigned long long height, unsigned long long length, unsigned long long ipitch, unsigned long long idist, unsigned long long opitch, unsigned long long odist);
template __global__ void copy_convert_kernel<u_int64_t>(u_int64_t *d_in, double *d_out, unsigned long long width, unsigned long long height, unsigned long long length, unsigned long long ipitch, unsigned long long idist, unsigned long long opitch, unsigned long long odist);
template __global__ void copy_convert_kernel<float>(float *d_in, double *d_out, unsigned long long width, unsigned long long height, unsigned long long length, unsigned long long ipitch, unsigned long long idist, unsigned long long opitch, unsigned long long odist);
template __global__ void copy_convert_kernel<double>(double *d_in, double *d_out, unsigned long long width, unsigned long long height, unsigned long long length, unsigned long long ipitch, unsigned long long idist, unsigned long long opitch, unsigned long long odist);

/*!
    Convert array from T to double on device and prepare for fft2 (single precision)
*/
template <typename T>
__global__ void copy_convert_single_kernel(T *d_in,
                                           float *d_out,
                                           unsigned long long width,
                                           unsigned long long height,
                                           unsigned long long length,
                                           unsigned long long ipitch,
                                           unsigned long long idist,
                                           unsigned long long opitch,
                                           unsigned long long odist)
{
    for (unsigned long long t = blockIdx.x; t < length; t += gridDim.x)
    {
        for (unsigned long long y = blockIdx.y; y < height; y += gridDim.y)
        {
            for (unsigned long long x = threadIdx.x; x < width; x += blockDim.x)
            {
                d_out[t * odist + y * opitch + x] = (float)d_in[t * idist + y * ipitch + x];
            }
        }
    }
}

template __global__ void copy_convert_single_kernel<u_int8_t>(u_int8_t *d_in, float *d_out, unsigned long long width, unsigned long long height, unsigned long long length, unsigned long long ipitch, unsigned long long idist, unsigned long long opitch, unsigned long long odist);
template __global__ void copy_convert_single_kernel<int16_t>(int16_t *d_in, float *d_out, unsigned long long width, unsigned long long height, unsigned long long length, unsigned long long ipitch, unsigned long long idist, unsigned long long opitch, unsigned long long odist);
template __global__ void copy_convert_single_kernel<u_int16_t>(u_int16_t *d_in, float *d_out, unsigned long long width, unsigned long long height, unsigned long long length, unsigned long long ipitch, unsigned long long idist, unsigned long long opitch, unsigned long long odist);
template __global__ void copy_convert_single_kernel<int32_t>(int32_t *d_in, float *d_out, unsigned long long width, unsigned long long height, unsigned long long length, unsigned long long ipitch, unsigned long long idist, unsigned long long opitch, unsigned long long odist);
template __global__ void copy_convert_single_kernel<u_int32_t>(u_int32_t *d_in, float *d_out, unsigned long long width, unsigned long long height, unsigned long long length, unsigned long long ipitch, unsigned long long idist, unsigned long long opitch, unsigned long long odist);
template __global__ void copy_convert_single_kernel<int64_t>(int64_t *d_in, float *d_out, unsigned long long width, unsigned long long height, unsigned long long length, unsigned long long ipitch, unsigned long long idist, unsigned long long opitch, unsigned long long odist);
template __global__ void copy_convert_single_kernel<u_int64_t>(u_int64_t *d_in, float *d_out, unsigned long long width, unsigned long long height, unsigned long long length, unsigned long long ipitch, unsigned long long idist, unsigned long long opitch, unsigned long long odist);
template __global__ void copy_convert_single_kernel<float>(float *d_in, float *d_out, unsigned long long width, unsigned long long height, unsigned long long length, unsigned long long ipitch, unsigned long long idist, unsigned long long opitch, unsigned long long odist);
template __global__ void copy_convert_single_kernel<double>(double *d_in, float *d_out, unsigned long long width, unsigned long long height, unsigned long long length, unsigned long long ipitch, unsigned long long idist, unsigned long long opitch, unsigned long long odist);

/*!
    Compute a *= A
 */
__global__ void scale_array_kernel(double2 *a,
                                   unsigned long long pitch,
                                   unsigned long long length,
                                   double A,
                                   unsigned long long N)
{
    for (unsigned long long row = blockIdx.x; row < N; row += gridDim.x)
    {
        for (unsigned long long tid = threadIdx.x; tid < length; tid += blockDim.x)
        {
            a[row * pitch + tid].x *= A;
            a[row * pitch + tid].y *= A;
        }
    }
}

/*!
    Compute a *= A
 */
__global__ void scale_array_single_kernel(float2 *a,
                                          unsigned long long pitch,
                                          unsigned long long length,
                                          float A,
                                          unsigned long long N)
{
    for (unsigned long long row = blockIdx.x; row < N; row += gridDim.x)
    {
        for (unsigned long long tid = threadIdx.x; tid < length; tid += blockDim.x)
        {
            a[row * pitch + tid].x *= A;
            a[row * pitch + tid].y *= A;
        }
    }
}

/*!
    Transpose complex matrix with pitch
 */
__global__ void transpose_complex_matrix_kernel(double2 *matIn,
                                                unsigned long long ipitch,
                                                double2 *matOut,
                                                unsigned long long opitch,
                                                unsigned long long width,
                                                unsigned long long height,
                                                unsigned long long NblocksX,
                                                unsigned long long NblocksY)
{
    __shared__ double2 tile[TILE_DIM][TILE_DIM + 1];

    for (unsigned long long blockIDX = blockIdx.x; blockIDX < NblocksX; blockIDX += gridDim.x)
    {
        for (unsigned long long blockIDY = blockIdx.y; blockIDY < NblocksY; blockIDY += gridDim.y)
        {
            unsigned long long i_x = blockIDX * TILE_DIM + threadIdx.x;
            unsigned long long i_y = blockIDY * TILE_DIM + threadIdx.y; // threadIdx.y goes from 0 to 7

            // load matrix portion into tile
            // every thread loads 4 elements into tile
            unsigned long long i;
            for (i = 0; i < TILE_DIM; i += BLOCK_ROWS)
            {
                if (i_x < width && (i_y + i) < height)
                {
                    tile[threadIdx.y + i][threadIdx.x] = matIn[(i_y + i) * ipitch + i_x];
                }
            }
            __syncthreads();

            // transpose block offset
            i_x = blockIDY * TILE_DIM + threadIdx.x;
            i_y = blockIDX * TILE_DIM + threadIdx.y;

            for (i = 0; i < TILE_DIM; i += BLOCK_ROWS)
            {
                if (i_x < height && (i_y + i) < width)
                {
                    matOut[(i_y + i) * opitch + i_x] = tile[threadIdx.x][threadIdx.y + i];
                }
            }
        }
    }
}

/*!
    Transpose complex matrix with pitch
 */
__global__ void transpose_complex_matrix_single_kernel(float2 *matIn,
                                                       unsigned long long ipitch,
                                                       float2 *matOut,
                                                       unsigned long long opitch,
                                                       unsigned long long width,
                                                       unsigned long long height,
                                                       unsigned long long NblocksX,
                                                       unsigned long long NblocksY)
{
    __shared__ float2 tile[TILE_DIM][TILE_DIM + 1];

    for (unsigned long long blockIDX = blockIdx.x; blockIDX < NblocksX; blockIDX += gridDim.x)
    {
        for (unsigned long long blockIDY = blockIdx.y; blockIDY < NblocksY; blockIDY += gridDim.y)
        {
            unsigned long long i_x = blockIDX * TILE_DIM + threadIdx.x;
            unsigned long long i_y = blockIDY * TILE_DIM + threadIdx.y; // threadIdx.y goes from 0 to 7

            // load matrix portion into tile
            // every thread loads 4 elements into tile
            unsigned long long i;
            for (i = 0; i < TILE_DIM; i += BLOCK_ROWS)
            {
                if (i_x < width && (i_y + i) < height)
                {
                    tile[threadIdx.y + i][threadIdx.x] = matIn[(i_y + i) * ipitch + i_x];
                }
            }
            __syncthreads();

            // transpose block offset
            i_x = blockIDY * TILE_DIM + threadIdx.x;
            i_y = blockIDX * TILE_DIM + threadIdx.y;

            for (i = 0; i < TILE_DIM; i += BLOCK_ROWS)
            {
                if (i_x < height && (i_y + i) < width)
                {
                    matOut[(i_y + i) * opitch + i_x] = tile[threadIdx.x][threadIdx.y + i];
                }
            }
        }
    }
}

/*!
    Compute structure function using differences.
    This is inspired by reduce6 from cuda samples.
 */
__global__ void structure_function_diff_kernel(double2 *d_in,
                                               double2 *d_out,
                                               unsigned int *d_lags,
                                               unsigned long long length,
                                               unsigned long long Nlags,
                                               unsigned long long Nq,
                                               unsigned long long pitch)
{
    // Handle to thread block group
    cg::thread_block cta = cg::this_thread_block();
    extern __shared__ double sdata[];
    unsigned long long blockSize = 2 * blockDim.x;
    unsigned long long tid = threadIdx.x;

    for (unsigned long long q = blockIdx.y; q < Nq; q += gridDim.y)
    {
        for (unsigned long long dt_idx = blockIdx.x; dt_idx < Nlags; dt_idx += gridDim.x)
        {
            unsigned long long dt = d_lags[dt_idx];

            // Perform first level of reduction
            // Reading from global memory, writing to shared memory
            double tmp_sum = 0.0;
            unsigned long long t = tid;
            while (t < length - dt)
            {
                double2 a = d_in[q * pitch + t];
                double2 b = d_in[q * pitch + t + dt];
                tmp_sum += (b.x - a.x) * (b.x - a.x) + (b.y - a.y) * (b.y - a.y);

                // ensure we don't read out of bounds
                if (t + blockDim.x < length - dt)
                {
                    double2 a = d_in[q * pitch + t + blockDim.x];
                    double2 b = d_in[q * pitch + t + blockDim.x + dt];
                    tmp_sum += (b.x - a.x) * (b.x - a.x) + (b.y - a.y) * (b.y - a.y);
                }

                t += blockSize;
            }

            // Each thread puts its local sum into shared memory
            sdata[tid] = tmp_sum;
            cg::sync(cta);

            // do reduction in shared mem
            if ((blockDim.x >= 512) && (tid < 256))
            {
                sdata[tid] = tmp_sum = tmp_sum + sdata[tid + 256];
            }

            cg::sync(cta);

            if ((blockDim.x >= 256) && (tid < 128))
            {
                sdata[tid] = tmp_sum = tmp_sum + sdata[tid + 128];
            }

            cg::sync(cta);

            if ((blockDim.x >= 128) && (tid < 64))
            {
                sdata[tid] = tmp_sum = tmp_sum + sdata[tid + 64];
            }

            cg::sync(cta);

            cg::thread_block_tile<32> tile32 = cg::tiled_partition<32>(cta);

            if (cta.thread_rank() < 32)
            {
                // Fetch final intermediate sum from 2nd warp
                if (blockDim.x >= 64)
                    tmp_sum += sdata[tid + 32];
                // Reduce final warp using shuffle
                for (int offset = tile32.size() / 2; offset > 0; offset /= 2)
                {
                    tmp_sum += tile32.shfl_down(tmp_sum, offset);
                }
            }

            // Write result for this block to global mem
            // Also normalize by number of occurrences
            // WARNING!!! IF WE ADD EXCLUDED FRAMES, WE NEED TO CHANGE THIS!!
            if (cta.thread_rank() == 0)
            {
                d_out[q * pitch + dt_idx].x = tmp_sum / double(length - dt);
            }
        }
    }
}

/*!
    Compute structure function using differences.
    This is inspired by reduce6 from cuda samples.
    Single precision.
 */
__global__ void structure_function_diff_single_kernel(float2 *d_in,
                                                      float2 *d_out,
                                                      unsigned int *d_lags,
                                                      unsigned long long length,
                                                      unsigned long long Nlags,
                                                      unsigned long long Nq,
                                                      unsigned long long pitch)
{
    // Handle to thread block group
    cg::thread_block cta = cg::this_thread_block();
    extern __shared__ double sdata[];
    unsigned long long blockSize = 2 * blockDim.x;
    unsigned long long tid = threadIdx.x;

    for (unsigned long long q = blockIdx.y; q < Nq; q += gridDim.y)
    {
        for (unsigned long long dt_idx = blockIdx.x; dt_idx < Nlags; dt_idx += gridDim.x)
        {
            unsigned long long dt = d_lags[dt_idx];

            // Perform first level of reduction
            // Reading from global memory, writing to shared memory
            double tmp_sum = 0.0;
            unsigned long long t = tid;
            while (t < length - dt)
            {
                double2 a = make_double2(d_in[q * pitch + t].x, d_in[q * pitch + t].y);
                double2 b = make_double2(d_in[q * pitch + t + dt].x, d_in[q * pitch + t + dt].y);
                tmp_sum += (b.x - a.x) * (b.x - a.x) + (b.y - a.y) * (b.y - a.y);

                // ensure we don't read out of bounds
                if (t + blockDim.x < length - dt)
                {
                    double2 a = make_double2(d_in[q * pitch + t + blockDim.x].x, d_in[q * pitch + t + blockDim.x].y);
                    double2 b = make_double2(d_in[q * pitch + t + blockDim.x + dt].x, d_in[q * pitch + t + blockDim.x + dt].y);
                    tmp_sum += (b.x - a.x) * (b.x - a.x) + (b.y - a.y) * (b.y - a.y);
                }

                t += blockSize;
            }

            // Each thread puts its local sum into shared memory
            sdata[tid] = tmp_sum;
            cg::sync(cta);

            // do reduction in shared mem
            if ((blockDim.x >= 512) && (tid < 256))
            {
                sdata[tid] = tmp_sum = tmp_sum + sdata[tid + 256];
            }

            cg::sync(cta);

            if ((blockDim.x >= 256) && (tid < 128))
            {
                sdata[tid] = tmp_sum = tmp_sum + sdata[tid + 128];
            }

            cg::sync(cta);

            if ((blockDim.x >= 128) && (tid < 64))
            {
                sdata[tid] = tmp_sum = tmp_sum + sdata[tid + 64];
            }

            cg::sync(cta);

            cg::thread_block_tile<32> tile32 = cg::tiled_partition<32>(cta);

            if (cta.thread_rank() < 32)
            {
                // Fetch final intermediate sum from 2nd warp
                if (blockDim.x >= 64)
                    tmp_sum += sdata[tid + 32];
                // Reduce final warp using shuffle
                for (int offset = tile32.size() / 2; offset > 0; offset /= 2)
                {
                    tmp_sum += tile32.shfl_down(tmp_sum, offset);
                }
            }

            // Write result for this block to global mem
            // Also normalize by number of occurrences
            // WARNING!!! IF WE ADD EXCLUDED FRAMES, WE NEED TO CHANGE THIS!!
            if (cta.thread_rank() == 0)
            {
                d_out[q * pitch + dt_idx].x = (float)(tmp_sum / double(length - dt));
            }
        }
    }
}

/*!
    Make full power spectrum (copy symmetric part)
*/
__global__ void make_full_powspec_kernel(double2 *d_in,
                                         unsigned long long ipitch,
                                         double *d_out,
                                         unsigned long long opitch,
                                         unsigned long long nxh,
                                         unsigned long long nx,
                                         unsigned long long ny,
                                         unsigned long long N,
                                         unsigned long long NblocksX,
                                         unsigned long long NblocksY)
{
    __shared__ double2 tile[TILE_DIM][TILE_DIM + 1];

    for (unsigned long long blockIDX = blockIdx.x; blockIDX < NblocksX; blockIDX += gridDim.x)
    {
        for (unsigned long long blockIDY = blockIdx.y; blockIDY < NblocksY; blockIDY += gridDim.y)
        {
            unsigned long long i_x = blockIDX * TILE_DIM + threadIdx.x;
            unsigned long long i_y = blockIDY * TILE_DIM + threadIdx.y; // threadIdx.y goes from 0 to 7

            // load matrix portion into tile
            // every thread loads 4 elements into tile
            unsigned long long i;
            for (i = 0; i < TILE_DIM; i += BLOCK_ROWS)
            {
                if (i_x < nxh && (i_y + i) < ny * N)
                {
                    tile[threadIdx.y + i][threadIdx.x] = d_in[(i_y + i) * ipitch + i_x];
                }
            }
            __syncthreads();

            unsigned long long curr_y;
            for (i = 0; i < TILE_DIM; i += BLOCK_ROWS)
            {
                if (i_x < nxh && (i_y + i) < ny * N)
                {
                    // copy real part (left side)
                    d_out[(i_y + i) * opitch + i_x] = tile[threadIdx.y + i][threadIdx.x].x;

                    // make symmetric part (right side)
                    if (i_x > 0)
                    {
                        curr_y = (i_y + i) % ny;
                        if (curr_y > 0)
                        {
                            d_out[(i_y + i + ny - 2 * curr_y) * opitch + nx - i_x] = tile[threadIdx.y + i][threadIdx.x].x;
                        }
                        else
                        {
                            d_out[(i_y + i) * opitch + nx - i_x] = tile[threadIdx.y + i][threadIdx.x].x;
                        }
                    }
                }
            }
        }
    }
}

/*!
    Make full power spectrum (copy symmetric part; single precision)
*/
__global__ void make_full_powspec_single_kernel(float2 *d_in,
                                                unsigned long long ipitch,
                                                float *d_out,
                                                unsigned long long opitch,
                                                unsigned long long nxh,
                                                unsigned long long nx,
                                                unsigned long long ny,
                                                unsigned long long N,
                                                unsigned long long NblocksX,
                                                unsigned long long NblocksY)
{
    __shared__ float2 tile[TILE_DIM][TILE_DIM + 1];

    for (unsigned long long blockIDX = blockIdx.x; blockIDX < NblocksX; blockIDX += gridDim.x)
    {
        for (unsigned long long blockIDY = blockIdx.y; blockIDY < NblocksY; blockIDY += gridDim.y)
        {
            unsigned long long i_x = blockIDX * TILE_DIM + threadIdx.x;
            unsigned long long i_y = blockIDY * TILE_DIM + threadIdx.y; // threadIdx.y goes from 0 to 7

            // load matrix portion into tile
            // every thread loads 4 elements into tile
            unsigned long long i;
            for (i = 0; i < TILE_DIM; i += BLOCK_ROWS)
            {
                if (i_x < nxh && (i_y + i) < ny * N)
                {
                    tile[threadIdx.y + i][threadIdx.x] = d_in[(i_y + i) * ipitch + i_x];
                }
            }
            __syncthreads();

            unsigned long long curr_y;
            for (i = 0; i < TILE_DIM; i += BLOCK_ROWS)
            {
                if (i_x < nxh && (i_y + i) < ny * N)
                {
                    // copy real part (left side)
                    d_out[(i_y + i) * opitch + i_x] = tile[threadIdx.y + i][threadIdx.x].x;

                    // make symmetric part (right side)
                    if (i_x > 0)
                    {
                        curr_y = (i_y + i) % ny;
                        if (curr_y > 0)
                        {
                            d_out[(i_y + i + ny - 2 * curr_y) * opitch + nx - i_x] = tile[threadIdx.y + i][threadIdx.x].x;
                        }
                        else
                        {
                            d_out[(i_y + i) * opitch + nx - i_x] = tile[threadIdx.y + i][threadIdx.x].x;
                        }
                    }
                }
            }
        }
    }
}

/*!
    Shift power spectrum
*/
__global__ void shift_powspec_kernel(double *d_in,
                                     unsigned long long ipitch,
                                     double *d_out,
                                     unsigned long long opitch,
                                     unsigned long long nx,
                                     unsigned long long ny,
                                     unsigned long long N,
                                     unsigned long long NblocksX,
                                     unsigned long long NblocksY)
{
    __shared__ double tile[TILE_DIM][TILE_DIM + 1];

    for (unsigned long long blockIDX = blockIdx.x; blockIDX < NblocksX; blockIDX += gridDim.x)
    {
        for (unsigned long long blockIDY = blockIdx.y; blockIDY < NblocksY; blockIDY += gridDim.y)
        {
            unsigned long long i_x = blockIDX * TILE_DIM + threadIdx.x;
            unsigned long long i_y = blockIDY * TILE_DIM + threadIdx.y; // threadIdx.y goes from 0 to 7

            // load matrix portion into tile
            // every thread loads 4 elements into tile
            unsigned long long i;
            for (i = 0; i < TILE_DIM; i += BLOCK_ROWS)
            {
                if (i_x < nx && (i_y + i) < ny * N)
                {
                    tile[threadIdx.y + i][threadIdx.x] = d_in[(i_y + i) * ipitch + i_x];
                }
            }
            __syncthreads();

            unsigned long long curr_y;
            unsigned long long shift_x, shift_y;
            for (i = 0; i < TILE_DIM; i += BLOCK_ROWS)
            {
                if (i_x < nx && (i_y + i) < ny * N)
                {
                    curr_y = (i_y + i) % ny;
                    shift_x = (i_x + nx / 2) % nx;
                    shift_y = (curr_y + ny / 2) % ny;
                    d_out[(i_y + i + shift_y - curr_y) * opitch + shift_x] = tile[threadIdx.y + i][threadIdx.x];
                }
            }
        }
    }
}

/*!
    Shift power spectrum (single precision)
*/
__global__ void shift_powspec_single_kernel(float *d_in,
                                            unsigned long long ipitch,
                                            float *d_out,
                                            unsigned long long opitch,
                                            unsigned long long nx,
                                            unsigned long long ny,
                                            unsigned long long N,
                                            unsigned long long NblocksX,
                                            unsigned long long NblocksY)
{
    __shared__ float tile[TILE_DIM][TILE_DIM + 1];

    for (unsigned long long blockIDX = blockIdx.x; blockIDX < NblocksX; blockIDX += gridDim.x)
    {
        for (unsigned long long blockIDY = blockIdx.y; blockIDY < NblocksY; blockIDY += gridDim.y)
        {
            unsigned long long i_x = blockIDX * TILE_DIM + threadIdx.x;
            unsigned long long i_y = blockIDY * TILE_DIM + threadIdx.y; // threadIdx.y goes from 0 to 7

            // load matrix portion into tile
            // every thread loads 4 elements into tile
            unsigned long long i;
            for (i = 0; i < TILE_DIM; i += BLOCK_ROWS)
            {
                if (i_x < nx && (i_y + i) < ny * N)
                {
                    tile[threadIdx.y + i][threadIdx.x] = d_in[(i_y + i) * ipitch + i_x];
                }
            }
            __syncthreads();

            unsigned long long curr_y;
            unsigned long long shift_x, shift_y;
            for (i = 0; i < TILE_DIM; i += BLOCK_ROWS)
            {
                if (i_x < nx && (i_y + i) < ny * N)
                {
                    curr_y = (i_y + i) % ny;
                    shift_x = (i_x + nx / 2) % nx;
                    shift_y = (curr_y + ny / 2) % ny;
                    d_out[(i_y + i + shift_y - curr_y) * opitch + shift_x] = tile[threadIdx.y + i][threadIdx.x];
                }
            }
        }
    }
}

/*!
    Compute square modulus of complex array
*/
__global__ void square_modulus_kernel(double2 *d_in,
                                      unsigned long long length,
                                      unsigned long long pitch,
                                      unsigned long long N)
{
    for (unsigned long long row = blockIdx.x; row < N; row += gridDim.x)
    {
        for (unsigned long long tid = threadIdx.x; tid < length; tid += blockDim.x)
        {
            unsigned long long i = row * pitch + tid;
            double2 a = d_in[i];
            d_in[i].x = a.x * a.x + a.y * a.y;
            d_in[i].y = 0.0;
        }
    }
}

/*!
    Compute square modulus of complex array (single precision)
*/
__global__ void square_modulus_single_kernel(float2 *d_in,
                                             unsigned long long length,
                                             unsigned long long pitch,
                                             unsigned long long N)
{
    for (unsigned long long row = blockIdx.x; row < N; row += gridDim.x)
    {
        for (unsigned long long tid = threadIdx.x; tid < length; tid += blockDim.x)
        {
            unsigned long long i = row * pitch + tid;
            float2 a = d_in[i];
            d_in[i].x = a.x * a.x + a.y * a.y;
            d_in[i].y = 0.0;
        }
    }
}

/*!
    Copy real part of element into imaginary part of opposite element
 */
__global__ void real2imagopposite_kernel(double2 *d_arr,
                                         unsigned long long length,
                                         unsigned long long pitch,
                                         unsigned long long N)
{
    for (unsigned long long row = blockIdx.x; row < N; row += gridDim.x)
    {
        for (unsigned long long tid = threadIdx.x; tid < length; tid += blockDim.x)
        {
            unsigned long long opp_idx = length - tid - 1;
            d_arr[row * pitch + tid].y = d_arr[row * pitch + opp_idx].x;
        }
    }
}

/*!
    Copy real part of element into imaginary part of opposite element (single precision)
 */
__global__ void real2imagopposite_single_kernel(float2 *d_arr,
                                                unsigned long long length,
                                                unsigned long long pitch,
                                                unsigned long long N)
{
    for (unsigned long long row = blockIdx.x; row < N; row += gridDim.x)
    {
        for (unsigned long long tid = threadIdx.x; tid < length; tid += blockDim.x)
        {
            unsigned long long opp_idx = length - tid - 1;
            d_arr[row * pitch + tid].y = d_arr[row * pitch + opp_idx].x;
        }
    }
}

/*!
    Do final linear combination c[i] = (a[0] - b[i].x - 2 * a[i]) / (length - i)
 */
__global__ void linear_combination_final_kernel(double2 *c,
                                                unsigned long long pitch_c,
                                                double2 *a,
                                                unsigned long long pitch_a,
                                                double2 *b,
                                                unsigned long long pitch_b,
                                                unsigned long long length,
                                                unsigned long long N)
{
    __shared__ double a0;

    for (unsigned long long blockID = blockIdx.x; blockID < N; blockID += gridDim.x)
    {
        unsigned long long blockOffset_a = blockID * pitch_a;
        unsigned long long blockOffset_b = blockID * pitch_b;
        unsigned long long blockOffset_c = blockID * pitch_c;

        // read first value of fft corr
        a0 = 2.0 * a[blockOffset_a].x;
        __syncthreads();

        for (unsigned long long threadID = threadIdx.x; threadID < length; threadID += blockDim.x)
        {
            double da = b[blockOffset_b + threadID].x;
            double dc = a[blockOffset_a + threadID].x;
            c[blockOffset_c + threadID].x = (a0 - da - 2 * dc) / (double)(length - threadID);
        }
    }
}

/*!
    Do final linear combination c[i] = (a[0] - b[i].x - 2 * a[i]) / (length - i) (single precision)
 */
__global__ void linear_combination_final_single_kernel(float2 *c,
                                                       unsigned long long pitch_c,
                                                       float2 *a,
                                                       unsigned long long pitch_a,
                                                       float2 *b,
                                                       unsigned long long pitch_b,
                                                       unsigned long long length,
                                                       unsigned long long N)
{
    __shared__ float a0;

    for (unsigned long long blockID = blockIdx.x; blockID < N; blockID += gridDim.x)
    {
        unsigned long long blockOffset_a = blockID * pitch_a;
        unsigned long long blockOffset_b = blockID * pitch_b;
        unsigned long long blockOffset_c = blockID * pitch_c;

        // read first value of fft corr
        a0 = 2.0 * a[blockOffset_a].x;
        __syncthreads();

        for (unsigned long long threadID = threadIdx.x; threadID < length; threadID += blockDim.x)
        {
            float da = b[blockOffset_b + threadID].x;
            float dc = a[blockOffset_a + threadID].x;
            c[blockOffset_c + threadID].x = (a0 - da - 2 * dc) / (float)(length - threadID);
        }
    }
}

/*!
    Keep only selected lags
*/
__global__ void copy_selected_lags_kernel(double2 *d_in,
                                          double2 *d_out,
                                          unsigned int *d_lags,
                                          unsigned long long Nlags,
                                          unsigned long long ipitch,
                                          unsigned long long opitch,
                                          unsigned long long N)
{
    for (unsigned long long blockID = blockIdx.x; blockID < N; blockID += gridDim.x)
    {
        unsigned long long blockOffsetIn = blockID * ipitch;
        unsigned long long blockOffsetOut = blockID * opitch;

        for (unsigned long long threadID = threadIdx.x; threadID < Nlags; threadID += blockDim.x)
        {
            d_out[blockOffsetOut + threadID] = d_in[blockOffsetIn + d_lags[threadID]];
        }
    }
}

/*!
    Keep only selected lags (single precision)
*/
__global__ void copy_selected_lags_single_kernel(float2 *d_in,
                                                 float2 *d_out,
                                                 unsigned int *d_lags,
                                                 unsigned long long Nlags,
                                                 unsigned long long ipitch,
                                                 unsigned long long opitch,
                                                 unsigned long long N)
{
    for (unsigned long long blockID = blockIdx.x; blockID < N; blockID += gridDim.x)
    {
        unsigned long long blockOffsetIn = blockID * ipitch;
        unsigned long long blockOffsetOut = blockID * opitch;

        for (unsigned long long threadID = threadIdx.x; threadID < Nlags; threadID += blockDim.x)
        {
            d_out[blockOffsetOut + threadID] = d_in[blockOffsetIn + d_lags[threadID]];
        }
    }
}

/*!
    Average power spectrum of input images
*/
__global__ void average_power_spectrum_kernel(double2 *d_in,
                                              double2 *d_out,
                                              unsigned long long length,
                                              unsigned long long pitch,
                                              unsigned long long Nq)
{
    // Handle to thread block group
    cg::thread_block cta = cg::this_thread_block();
    extern __shared__ double sdata[];
    unsigned long long blockSize = 2 * blockDim.x;
    unsigned long long tid = threadIdx.x;

    for (unsigned long long q = blockIdx.x; q < Nq; q += gridDim.x)
    {
        double tmp_sum = 0.0;
        for (unsigned long long t = tid; t < length; t += blockSize)
        {
            double2 a = d_in[q * pitch + t];
            tmp_sum += a.x * a.x + a.y * a.y;

            // ensure we don't read out of bounds
            if (t + blockDim.x < length)
            {
                a = d_in[q * pitch + t + blockDim.x];
                tmp_sum += a.x * a.x + a.y * a.y;
            }
        }

        // Each thread puts its local sum into shared memory
        sdata[tid] = tmp_sum;
        cg::sync(cta);

        // do reduction in shared mem
        if ((blockDim.x >= 512) && (tid < 256))
        {
            sdata[tid] = tmp_sum = tmp_sum + sdata[tid + 256];
        }

        cg::sync(cta);

        if ((blockDim.x >= 256) && (tid < 128))
        {
            sdata[tid] = tmp_sum = tmp_sum + sdata[tid + 128];
        }

        cg::sync(cta);

        if ((blockDim.x >= 128) && (tid < 64))
        {
            sdata[tid] = tmp_sum = tmp_sum + sdata[tid + 64];
        }

        cg::sync(cta);

        cg::thread_block_tile<32> tile32 = cg::tiled_partition<32>(cta);

        if (cta.thread_rank() < 32)
        {
            // Fetch final intermediate sum from 2nd warp
            if (blockDim.x >= 64)
                tmp_sum += sdata[tid + 32];
            // Reduce final warp using shuffle
            for (int offset = tile32.size() / 2; offset > 0; offset /= 2)
            {
                tmp_sum += tile32.shfl_down(tmp_sum, offset);
            }
        }

        // Write result for this block to global mem
        // Also normalize by number of frames
        // WARNING!!! IF WE ADD EXCLUDED FRAMES, WE NEED TO CHANGE THIS!!
        if (cta.thread_rank() == 0)
        {
            d_out[q].x = tmp_sum / (double)length;
        }
    }
}

/*!
    Average power spectrum of input images (single precision)
*/
__global__ void average_power_spectrum_single_kernel(float2 *d_in,
                                                     float2 *d_out,
                                                     unsigned long long length,
                                                     unsigned long long pitch,
                                                     unsigned long long Nq)
{
    // Handle to thread block group
    cg::thread_block cta = cg::this_thread_block();
    extern __shared__ double sdata[];
    unsigned long long blockSize = 2 * blockDim.x;
    unsigned long long tid = threadIdx.x;

    for (unsigned long long q = blockIdx.x; q < Nq; q += gridDim.x)
    {
        double tmp_sum = 0.0;
        for (unsigned long long t = tid; t < length; t += blockSize)
        {
            double2 a = make_double2(d_in[q * pitch + t].x, d_in[q * pitch + t].y);
            tmp_sum += a.x * a.x + a.y * a.y;

            // ensure we don't read out of bounds
            if (t + blockDim.x < length)
            {
                a = make_double2(d_in[q * pitch + t + blockDim.x].x, d_in[q * pitch + t + blockDim.x].y);
                tmp_sum += a.x * a.x + a.y * a.y;
            }
        }

        // Each thread puts its local sum into shared memory
        sdata[tid] = tmp_sum;
        cg::sync(cta);

        // do reduction in shared mem
        if ((blockDim.x >= 512) && (tid < 256))
        {
            sdata[tid] = tmp_sum = tmp_sum + sdata[tid + 256];
        }

        cg::sync(cta);

        if ((blockDim.x >= 256) && (tid < 128))
        {
            sdata[tid] = tmp_sum = tmp_sum + sdata[tid + 128];
        }

        cg::sync(cta);

        if ((blockDim.x >= 128) && (tid < 64))
        {
            sdata[tid] = tmp_sum = tmp_sum + sdata[tid + 64];
        }

        cg::sync(cta);

        cg::thread_block_tile<32> tile32 = cg::tiled_partition<32>(cta);

        if (cta.thread_rank() < 32)
        {
            // Fetch final intermediate sum from 2nd warp
            if (blockDim.x >= 64)
                tmp_sum += sdata[tid + 32];
            // Reduce final warp using shuffle
            for (int offset = tile32.size() / 2; offset > 0; offset /= 2)
            {
                tmp_sum += tile32.shfl_down(tmp_sum, offset);
            }
        }

        // Write result for this block to global mem
        // Also normalize by number of frames
        // WARNING!!! IF WE ADD EXCLUDED FRAMES, WE NEED TO CHANGE THIS!!
        if (cta.thread_rank() == 0)
        {
            d_out[q].x = (float)(tmp_sum / (double)length);
        }
    }
}

/*!
    Average over time of Fourier transformed input images
*/
__global__ void average_complex_kernel(double2 *d_in,
                                       double2 *d_out,
                                       unsigned long long length,
                                       unsigned long long pitch,
                                       unsigned long long Nq)
{
    // Handle to thread block group
    cg::thread_block cta = cg::this_thread_block();
    extern __shared__ double2 sdata2[];
    unsigned long long blockSize = 2 * blockDim.x;
    unsigned long long tid = threadIdx.x;

    for (unsigned long long q = blockIdx.x; q < Nq; q += gridDim.x)
    {
        double tmp_sum_r = 0.0;
        double tmp_sum_i = 0.0;
        for (unsigned long long t = tid; t < length; t += blockSize)
        {
            double2 a = d_in[q * pitch + t];
            tmp_sum_r += a.x;
            tmp_sum_i += a.y;

            // Ensure we don't read out of bounds
            if (t + blockDim.x < length)
            {
                a = d_in[q * pitch + t + blockDim.x];
                tmp_sum_r += a.x;
                tmp_sum_i += a.y;
            }
        }

        // Each thread puts its local sum into shared memory
        sdata2[tid].x = tmp_sum_r;
        sdata2[tid].y = tmp_sum_i;
        cg::sync(cta);

        // do reduction in shared mem
        if ((blockDim.x >= 512) && (tid < 256))
        {
            sdata2[tid].x = tmp_sum_r = tmp_sum_r + sdata2[tid + 256].x;
            sdata2[tid].y = tmp_sum_i = tmp_sum_i + sdata2[tid + 256].y;
        }

        cg::sync(cta);

        if ((blockDim.x >= 256) && (tid < 128))
        {
            sdata2[tid].x = tmp_sum_r = tmp_sum_r + sdata2[tid + 128].x;
            sdata2[tid].y = tmp_sum_i = tmp_sum_i + sdata2[tid + 128].y;
        }

        cg::sync(cta);

        if ((blockDim.x >= 128) && (tid < 64))
        {
            sdata2[tid].x = tmp_sum_r = tmp_sum_r + sdata2[tid + 64].x;
            sdata2[tid].y = tmp_sum_i = tmp_sum_i + sdata2[tid + 64].y;
        }

        cg::sync(cta);

        cg::thread_block_tile<32> tile32 = cg::tiled_partition<32>(cta);

        if (cta.thread_rank() < 32)
        {
            // Fetch final intermediate sum from 2nd warp
            if (blockDim.x >= 64)
                tmp_sum_r += sdata2[tid + 32].x;
            tmp_sum_i += sdata2[tid + 32].y;
            // Reduce final warp using shuffle
            for (int offset = tile32.size() / 2; offset > 0; offset /= 2)
            {
                tmp_sum_r += tile32.shfl_down(tmp_sum_r, offset);
                tmp_sum_i += tile32.shfl_down(tmp_sum_i, offset);
            }
        }

        // Write result for this block to global mem
        // Also normalize by number of frames
        // WARNING!!! IF WE ADD EXCLUDED FRAMES, WE NEED TO CHANGE THIS!!
        if (cta.thread_rank() == 0)
        {
            d_out[q].x = tmp_sum_r / (double)length;
            d_out[q].y = tmp_sum_i / (double)length;
        }
    }
}

/*!
    Average over time of Fourier transformed input images (single precision)
*/
__global__ void average_complex_single_kernel(float2 *d_in,
                                              float2 *d_out,
                                              unsigned long long length,
                                              unsigned long long pitch,
                                              unsigned long long Nq)
{
    // Handle to thread block group
    cg::thread_block cta = cg::this_thread_block();
    extern __shared__ double2 sdata2[];
    unsigned long long blockSize = 2 * blockDim.x;
    unsigned long long tid = threadIdx.x;

    for (unsigned long long q = blockIdx.x; q < Nq; q += gridDim.x)
    {
        double tmp_sum_r = 0.0;
        double tmp_sum_i = 0.0;
        for (unsigned long long t = tid; t < length; t += blockSize)
        {
            double2 a = make_double2(d_in[q * pitch + t].x, d_in[q * pitch + t].y);
            tmp_sum_r += a.x;
            tmp_sum_i += a.y;

            // Ensure we don't read out of bounds
            if (t + blockDim.x < length)
            {
                a = make_double2(d_in[q * pitch + t + blockDim.x].x, d_in[q * pitch + t + blockDim.x].y);
                tmp_sum_r += a.x;
                tmp_sum_i += a.y;
            }
        }

        // Each thread puts its local sum into shared memory
        sdata2[tid].x = tmp_sum_r;
        sdata2[tid].y = tmp_sum_i;
        cg::sync(cta);

        // do reduction in shared mem
        if ((blockDim.x >= 512) && (tid < 256))
        {
            sdata2[tid].x = tmp_sum_r = tmp_sum_r + sdata2[tid + 256].x;
            sdata2[tid].y = tmp_sum_i = tmp_sum_i + sdata2[tid + 256].y;
        }

        cg::sync(cta);

        if ((blockDim.x >= 256) && (tid < 128))
        {
            sdata2[tid].x = tmp_sum_r = tmp_sum_r + sdata2[tid + 128].x;
            sdata2[tid].y = tmp_sum_i = tmp_sum_i + sdata2[tid + 128].y;
        }

        cg::sync(cta);

        if ((blockDim.x >= 128) && (tid < 64))
        {
            sdata2[tid].x = tmp_sum_r = tmp_sum_r + sdata2[tid + 64].x;
            sdata2[tid].y = tmp_sum_i = tmp_sum_i + sdata2[tid + 64].y;
        }

        cg::sync(cta);

        cg::thread_block_tile<32> tile32 = cg::tiled_partition<32>(cta);

        if (cta.thread_rank() < 32)
        {
            // Fetch final intermediate sum from 2nd warp
            if (blockDim.x >= 64)
                tmp_sum_r += sdata2[tid + 32].x;
            tmp_sum_i += sdata2[tid + 32].y;
            // Reduce final warp using shuffle
            for (int offset = tile32.size() / 2; offset > 0; offset /= 2)
            {
                tmp_sum_r += tile32.shfl_down(tmp_sum_r, offset);
                tmp_sum_i += tile32.shfl_down(tmp_sum_i, offset);
            }
        }

        // Write result for this block to global mem
        // Also normalize by number of frames
        // WARNING!!! IF WE ADD EXCLUDED FRAMES, WE NEED TO CHANGE THIS!!
        if (cta.thread_rank() == 0)
        {
            d_out[q] = make_float2(tmp_sum_r / (double)length, tmp_sum_i / (double)length);
        }
    }
}

/*!
    Linear combination c = A * a + B * b
*/
__global__ void linear_combination_kernel(double2 *c,
                                          double2 *a,
                                          double2 A,
                                          double2 *b,
                                          double2 B,
                                          unsigned int N)
{
    unsigned int tid = blockDim.x * blockIdx.x + threadIdx.x;

    if (tid < N)
    {
        c[tid].x = A.x * a[tid].x + B.x * b[tid].x;
        c[tid].y = A.y * a[tid].y + B.y * b[tid].y;
    }
}

/*!
    Linear combination c = A * a + B * b (single precision)
*/
__global__ void linear_combination_single_kernel(float2 *c,
                                                 float2 *a,
                                                 float2 A,
                                                 float2 *b,
                                                 float2 B,
                                                 unsigned int N)
{
    unsigned int tid = blockDim.x * blockIdx.x + threadIdx.x;

    if (tid < N)
    {
        c[tid].x = A.x * a[tid].x + B.x * b[tid].x;
        c[tid].y = A.y * a[tid].y + B.y * b[tid].y;
    }
}
