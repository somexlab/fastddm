#include "hip/hip_runtime.h"
// Maintainer: enrico-lattuada

/*! \file ddm_cuda.cu
    \brief Definition of core CUDA Differential Dynamic Microscopy functions
*/

// *** headers ***
#include "ddm_cuda.cuh"

#include "helper_debug.cuh"
#include "helper_cufft.cuh"
#include "helper_ddm_cuda.cuh"
#include "helper_prefix_sum.cuh"

#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>

#include <stdlib.h>

// #include <chrono>
// using namespace std::chrono;
// typedef std::chrono::high_resolution_clock Time;
// typedef std::chrono::duration<float> fsec;

// auto t0 = Time::now();
// auto t1 = Time::now();
// fsec fs = t0 - t0;
// fprintf(stdout, "%f\n", fs.count());

#ifndef SINGLE_PRECISION
typedef hipfftDoubleComplex CUFFTCOMPLEX;
typedef double2 Scalar2;
hipfftResult (*CufftExecR2C)(hipfftHandle, hipfftDoubleReal*, hipfftDoubleComplex*) = &hipfftExecD2Z;
hipfftResult (*CufftExecC2C)(hipfftHandle, hipfftDoubleComplex*, hipfftDoubleComplex*, int) = &hipfftExecZ2Z;
double2 (*make_Scalar2)(double, double) = &make_double2;
#else
typedef hipfftComplex CUFFTCOMPLEX;
typedef float2 Scalar2;
hipfftResult (*CufftExecR2C)(hipfftHandle, hipfftReal*, hipfftComplex*) = &hipfftExecR2C;
hipfftResult (*CufftExecC2C)(hipfftHandle, hipfftComplex*, hipfftComplex*, int) = &hipfftExecC2C;
float2 (*make_Scalar2)(float, float) = &make_float2;
#endif

// *** code ***
const unsigned long long TILE_DIM = 32;  // leave this unchanged!! (tile dimension for matrix transpose)
const unsigned long long BLOCK_ROWS = 8; // leave this unchanged!! (block rows for matrix transpose)

/*!
    Transfer images on GPU and compute fft2
 */
template <typename T>
void compute_fft2(const T *h_in,
                  Scalar *h_out,
                  unsigned long long width,
                  unsigned long long height,
                  unsigned long long length,
                  unsigned long long nx,
                  unsigned long long ny,
                  unsigned long long num_fft2,
                  unsigned long long buff_pitch,
                  unsigned long long pitch_nx)
{
    int device_id;
    hipGetDevice(&device_id);

    // compute half width of fft2
    unsigned long long _nx = nx / 2 + 1;
    // compute batch number of fft2
    unsigned long long batch = (length - 1) / num_fft2 + 1;
    // compute fft2 normalizaton factor
    double norm_fact = 1.0 / sqrt((double)(nx * ny));

    // ***Allocate device arrays
    // workspace
    Scalar *d_workspace;
    gpuErrchk(hipMalloc(&d_workspace, 2 * pitch_nx * ny * batch * sizeof(Scalar)));
    // buffer (only allocate if T is not double)
    T *d_buff;
    if (!std::is_same<T, Scalar>::value)
    {
        gpuErrchk(hipMalloc(&d_buff, buff_pitch * height * batch * sizeof(T)));
    }

    // ***Create fft2 plan
    hipfftHandle fft2_plan = fft2_create_plan(nx,
                                             ny,
                                             batch,
                                             pitch_nx);

    // Compute efficient execution configuration
    int numSMs; // Number of streaming multiprocessors
    gpuErrchk(hipDeviceGetAttribute(&numSMs, hipDeviceAttributeMultiprocessorCount, device_id));
    int maxGridSizeX, maxGridSizeY;
    gpuErrchk(hipDeviceGetAttribute(&maxGridSizeX, hipDeviceAttributeMaxGridDimX, device_id));
    gpuErrchk(hipDeviceGetAttribute(&maxGridSizeY, hipDeviceAttributeMaxGridDimY, device_id));

    // copy_convert_kernel
    int blockSize_copy; // The launch configurator returned block size
    int minGridSize;    // The minimum grid size needed to achieve the
                        // maximum occupancy for a full device launch
    gpuErrchk(hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize_copy, copy_convert_kernel<T>, 0, 0));
    dim3 gridSize_copy(min(1ULL * maxGridSizeX, batch), min(1ULL * maxGridSizeY, height), 1);

    // scale kernel
    int blockSize_scale; // The launch configurator returned block size
    int gridSize_scale;  // The actual grid size needed, based on input size

    gpuErrchk(hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize_scale, scale_array_kernel, 0, 0));
    // Round up according to array size
    gridSize_scale = min((ny * batch + blockSize_scale - 1) / blockSize_scale, 32ULL * numSMs);

    // ***Batched fft2
    for (unsigned long long ii = 0; ii < num_fft2; ii++)
    {
        // rezero workspace array
        gpuErrchk(hipMemset(d_workspace, 0.0, 2 * pitch_nx * ny * batch * sizeof(Scalar)));

        // ***Copy values to device
        if (std::is_same<T, Scalar>::value)
        {
            // copy values directly to workspace with zero padding
            // number of images to copy
            unsigned long long num_imgs_copy = (ii + 1) * batch > length ? length - ii * batch : batch;
            // use hipMemcpy3D
            hipMemcpy3DParms params = {0};
            params.srcArray = NULL;
            params.srcPos = make_hipPos(0, 0, ii * batch);
            params.srcPtr = make_hipPitchedPtr((Scalar *)h_in, width * sizeof(Scalar), width, height);
            params.dstArray = NULL;
            params.dstPos = make_hipPos(0, 0, 0);
            params.dstPtr = make_hipPitchedPtr(d_workspace, 2 * pitch_nx * sizeof(Scalar), 2 * pitch_nx, ny);
            params.extent = make_hipExtent(width * sizeof(Scalar), height, num_imgs_copy);
            params.kind = hipMemcpyHostToDevice;

            gpuErrchk(hipMemcpy3D(&params));
        }
        else
        {
            // rezero buffer array
            gpuErrchk(hipMemset(d_buff, (T)0, buff_pitch * height * batch * sizeof(T)));

            // offset index
            unsigned long long offset = ii * width * height * batch;
            // number of rows to copy
            unsigned long long num_rows_copy = (ii + 1) * batch > length ? height * (length - ii * batch) : height * batch;
            // copy values to buffer
            gpuErrchk(hipMemcpy2D(d_buff, buff_pitch * sizeof(T), h_in + offset, width * sizeof(T), width * sizeof(T), num_rows_copy, hipMemcpyHostToDevice));

            // convert values of buffer into workspace
            copy_convert_kernel<<<gridSize_copy, blockSize_copy>>>(d_buff,
                                                                   d_workspace,
                                                                   width,
                                                                   height,
                                                                   batch,
                                                                   buff_pitch,
                                                                   buff_pitch * height,
                                                                   2 * pitch_nx,
                                                                   2 * pitch_nx * ny);
            gpuErrchk(hipPeekAtLastError());
            gpuErrchk(hipDeviceSynchronize());
        }

        // ***Execute fft2 plan
        cufftSafeCall(CufftExecR2C(fft2_plan, d_workspace, (CUFFTCOMPLEX *)d_workspace));
        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipDeviceSynchronize());

        // ***Normalize fft2
        // Starting index
        unsigned long long start = ii * ny * batch;
        // Final index (if exceeds array size, truncate)
        unsigned long long end = (ii + 1) * batch > length ? length * ny : (ii + 1) * ny * batch;
        // scale array
        scale_array_kernel<<<gridSize_scale, blockSize_scale>>>((Scalar2 *)d_workspace,
                                                                pitch_nx,
                                                                _nx,
                                                                norm_fact,
                                                                end - start);
        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipDeviceSynchronize());

        // ***Copy values back to host
        gpuErrchk(hipMemcpy2D((Scalar2 *)h_out + _nx * start, _nx * sizeof(Scalar2), (Scalar2 *)d_workspace, pitch_nx * sizeof(Scalar2), _nx * sizeof(Scalar2), end - start, hipMemcpyDeviceToHost));
    }

    // ***Free memory
    gpuErrchk(hipFree(d_workspace));
    gpuErrchk(hipFree(d_buff));
    cufftSafeCall(hipfftDestroy(fft2_plan));
}

template void compute_fft2<u_int8_t>(const u_int8_t *h_in, Scalar *h_out, unsigned long long width, unsigned long long height, unsigned long long length, unsigned long long nx, unsigned long long ny, unsigned long long num_fft2, unsigned long long buff_pitch, unsigned long long pitch_nx);
template void compute_fft2<int16_t>(const int16_t *h_in, Scalar *h_out, unsigned long long width, unsigned long long height, unsigned long long length, unsigned long long nx, unsigned long long ny, unsigned long long num_fft2, unsigned long long buff_pitch, unsigned long long pitch_nx);
template void compute_fft2<u_int16_t>(const u_int16_t *h_in, Scalar *h_out, unsigned long long width, unsigned long long height, unsigned long long length, unsigned long long nx, unsigned long long ny, unsigned long long num_fft2, unsigned long long buff_pitch, unsigned long long pitch_nx);
template void compute_fft2<int32_t>(const int32_t *h_in, Scalar *h_out, unsigned long long width, unsigned long long height, unsigned long long length, unsigned long long nx, unsigned long long ny, unsigned long long num_fft2, unsigned long long buff_pitch, unsigned long long pitch_nx);
template void compute_fft2<u_int32_t>(const u_int32_t *h_in, Scalar *h_out, unsigned long long width, unsigned long long height, unsigned long long length, unsigned long long nx, unsigned long long ny, unsigned long long num_fft2, unsigned long long buff_pitch, unsigned long long pitch_nx);
template void compute_fft2<int64_t>(const int64_t *h_in, Scalar *h_out, unsigned long long width, unsigned long long height, unsigned long long length, unsigned long long nx, unsigned long long ny, unsigned long long num_fft2, unsigned long long buff_pitch, unsigned long long pitch_nx);
template void compute_fft2<u_int64_t>(const u_int64_t *h_in, Scalar *h_out, unsigned long long width, unsigned long long height, unsigned long long length, unsigned long long nx, unsigned long long ny, unsigned long long num_fft2, unsigned long long buff_pitch, unsigned long long pitch_nx);
template void compute_fft2<float>(const float *h_in, Scalar *h_out, unsigned long long width, unsigned long long height, unsigned long long length, unsigned long long nx, unsigned long long ny, unsigned long long num_fft2, unsigned long long buff_pitch, unsigned long long pitch_nx);
template void compute_fft2<double>(const double *h_in, Scalar *h_out, unsigned long long width, unsigned long long height, unsigned long long length, unsigned long long nx, unsigned long long ny, unsigned long long num_fft2, unsigned long long buff_pitch, unsigned long long pitch_nx);

/*!
    Compute image structure function using differences on the GPU
 */
void structure_function_diff(Scalar *h_in,
                             vector<unsigned int> lags,
                             unsigned long long length,
                             unsigned long long nx,
                             unsigned long long ny,
                             unsigned long long num_chunks,
                             unsigned long long pitch_q,
                             unsigned long long pitch_t)
{
    int device_id;
    hipGetDevice(&device_id);

    unsigned long long _nx = nx / 2 + 1;                             // fft2 r2c number of complex elements over x
    unsigned long long chunk_size = (_nx * ny - 1) / num_chunks + 1; // number of q points in a chunk

    // ***Allocate space on device
    // workspaces
    Scalar *d_workspace1, *d_workspace2;
    unsigned long long workspace_size = max(pitch_q * length, chunk_size * pitch_t) * 2 * sizeof(Scalar);
    gpuErrchk(hipMalloc(&d_workspace1, workspace_size));
    gpuErrchk(hipMalloc(&d_workspace2, workspace_size));
    // helper arrays
    unsigned int *d_lags;
    Scalar2 *d_power_spec, *d_var;
    gpuErrchk(hipMalloc(&d_lags, lags.size() * sizeof(unsigned int)));
    gpuErrchk(hipMemcpy(d_lags, lags.data(), lags.size() * sizeof(unsigned int), hipMemcpyHostToDevice));
    gpuErrchk(hipMalloc(&d_power_spec, chunk_size * sizeof(Scalar2)));
    gpuErrchk(hipMalloc(&d_var, chunk_size * sizeof(Scalar2)));

    // ***Compute optimal kernels execution parameters
    // transpose_complex_matrix_kernel
    dim3 blockSize_tran(TILE_DIM, BLOCK_ROWS, 1);
    int maxGridSizeX, maxGridSizeY;
    gpuErrchk(hipDeviceGetAttribute(&maxGridSizeX, hipDeviceAttributeMaxGridDimX, device_id));
    gpuErrchk(hipDeviceGetAttribute(&maxGridSizeY, hipDeviceAttributeMaxGridDimY, device_id));
    int gridSize_tran1_x = min((chunk_size + TILE_DIM - 1) / TILE_DIM, (unsigned long long)maxGridSizeX);
    int gridSize_tran1_y = min((length + TILE_DIM - 1) / TILE_DIM, (unsigned long long)maxGridSizeY);
    dim3 gridSize_tran1(gridSize_tran1_x, gridSize_tran1_y, 1);
    int gridSize_tran2_x = min(((unsigned long long)(lags.size()) + TILE_DIM - 1) / TILE_DIM, (unsigned long long)maxGridSizeX);
    int gridSize_tran2_y = min((chunk_size + TILE_DIM - 1) / TILE_DIM, (unsigned long long)maxGridSizeY);
    dim3 gridSize_tran2(gridSize_tran2_x, gridSize_tran2_y, 1);

    // correlation part
    int blockSize_corr = min(nextPowerOfTwo(length), 512ULL);
    int gridSize_corr_x = min(((unsigned long long)(lags.size()) + blockSize_corr - 1) / blockSize_corr, (unsigned long long)maxGridSizeX);
    int gridSize_corr_y = min(chunk_size, (unsigned long long)maxGridSizeY);
    dim3 gridSize_corr(gridSize_corr_x, gridSize_corr_y, 1);
    int smemSize = (blockSize_corr <= 32) ? 2ULL * blockSize_corr * sizeof(double) : 1ULL * blockSize_corr * sizeof(double);

    // power spectrum and variance (reduction)
    int minGridSize; // The minimum grid size needed to achieve the
                     // maximum occupancy for a full device launch
    int numSMs;      // Number of streaming multiprocessors
    gpuErrchk(hipDeviceGetAttribute(&numSMs, hipDeviceAttributeMultiprocessorCount, device_id));

    int blockSize_red = min(nextPowerOfTwo(length), 512ULL);
    int gridSize_red = min(chunk_size, (unsigned long long)maxGridSizeX);
    int smemSize2 = (blockSize_corr <= 32) ? 2ULL * blockSize_corr * sizeof(double2) : 1ULL * blockSize_corr * sizeof(double2);

    // linear combination for variance
    int blockSize_lc; // The launch configurator returned block size
    int gridSize_lc;  // The actual grid size needed, based on input size
    gpuErrchk(hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize_lc, linear_combination_kernel, 0, 0));
    // Round up according to array size
    gridSize_lc = min((chunk_size + blockSize_lc - 1) / blockSize_lc, 32ULL * numSMs);

    // ***Loop over chunks
    for (unsigned long long chunk = 0; chunk < num_chunks; chunk++)
    {
        // ***Get start and end q values
        unsigned long long q_start = chunk * chunk_size;
        unsigned long long q_end = (chunk + 1) * chunk_size < _nx * ny ? (chunk + 1) * chunk_size : _nx * ny;
        unsigned long long curr_chunk_size = q_end - q_start;

        if (curr_chunk_size != chunk_size)
        {
            // if chunk size changes, modify kernel execution parameters for transpose
            gridSize_tran1.x = min((curr_chunk_size + TILE_DIM - 1) / TILE_DIM, (unsigned long long)maxGridSizeX);
            gridSize_tran2.y = min((curr_chunk_size + TILE_DIM - 1) / TILE_DIM, (unsigned long long)maxGridSizeY);
            gridSize_corr.y = min(curr_chunk_size, (unsigned long long)maxGridSizeY);
        }

        // ***Copy values from host to device
        // elements are complex
        // to speed up transfer, use pitch_q
        gpuErrchk(hipMemcpy2D(d_workspace2,
                               2 * pitch_q * sizeof(Scalar),
                               h_in + 2 * q_start,
                               2 * _nx * ny * sizeof(Scalar),
                               2 * curr_chunk_size * sizeof(Scalar),
                               length,
                               hipMemcpyHostToDevice));

        // ***Transpose array (d_workspace2 --> d_workspace1)
        transpose_complex_matrix_kernel<<<gridSize_tran1, blockSize_tran>>>((Scalar2 *)d_workspace2,
                                                                            pitch_q,
                                                                            (Scalar2 *)d_workspace1,
                                                                            pitch_t,
                                                                            curr_chunk_size,
                                                                            length,
                                                                            (curr_chunk_size + TILE_DIM - 1) / TILE_DIM,
                                                                            (length + TILE_DIM - 1) / TILE_DIM);
        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipDeviceSynchronize());

        // ***Zero-out workspace2
        gpuErrchk(hipMemset(d_workspace2, 0.0, workspace_size));

        // ***Compute structure function using differences (d_workspace1 --> d_workspace2)
        structure_function_diff_kernel<<<gridSize_corr, blockSize_corr, smemSize>>>((Scalar2 *)d_workspace1,
                                                                                    (Scalar2 *)d_workspace2,
                                                                                    d_lags,
                                                                                    length,
                                                                                    lags.size(),
                                                                                    curr_chunk_size,
                                                                                    pitch_t);
        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipDeviceSynchronize());

        // ***Compute power spectrum (d_workspace1 --> d_power_spect)
        average_power_spectrum_kernel<<<gridSize_red, blockSize_red, smemSize>>>((Scalar2 *)d_workspace1,
                                                                                 d_power_spec,
                                                                                 length,
                                                                                 pitch_t,
                                                                                 curr_chunk_size);

        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipDeviceSynchronize());

        // ***Compute variance (d_workspace1 --> d_var)
        // compute average over time
        average_complex_kernel<<<gridSize_red, blockSize_red, smemSize2>>>((Scalar2 *)d_workspace1,
                                                                           d_var,
                                                                           length,
                                                                           pitch_t,
                                                                           curr_chunk_size);

        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipDeviceSynchronize());

        // compute square modulus
        square_modulus_kernel<<<gridSize_lc, blockSize_lc>>>(d_var,
                                                             1,
                                                             1,
                                                             curr_chunk_size);

        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipDeviceSynchronize());

        // linear combination (d_var = d_power_spec - d_var)
        linear_combination_kernel<<<gridSize_lc, blockSize_lc>>>(d_var,
                                                                 d_power_spec,
                                                                 make_double2(1.0, 0.0),
                                                                 d_var,
                                                                 make_double2(-1.0, 0.0),
                                                                 curr_chunk_size);

        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipDeviceSynchronize());

        // ***Transpose array (d_workspace2 --> d_workspace1)
        transpose_complex_matrix_kernel<<<gridSize_tran2, blockSize_tran>>>((Scalar2 *)d_workspace2,
                                                                            pitch_t,
                                                                            (Scalar2 *)d_workspace1,
                                                                            pitch_q,
                                                                            lags.size(),
                                                                            curr_chunk_size,
                                                                            (lags.size() + TILE_DIM - 1) / TILE_DIM,
                                                                            (curr_chunk_size + TILE_DIM - 1) / TILE_DIM);
        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipDeviceSynchronize());

        // ***Copy values from device to host
        // elements are treated as complex
        // to speed up transfer, use pitch_q
        gpuErrchk(hipMemcpy2D(h_in + 2 * q_start,
                               2 * _nx * ny * sizeof(Scalar),
                               d_workspace1,
                               2 * pitch_q * sizeof(Scalar),
                               2 * curr_chunk_size * sizeof(Scalar),
                               lags.size(),
                               hipMemcpyDeviceToHost));
        // copy power spectrum
        gpuErrchk(hipMemcpy((Scalar2 *)h_in + (unsigned long long)(lags.size()) * _nx * ny + q_start,
                             d_power_spec,
                             curr_chunk_size * sizeof(Scalar2),
                             hipMemcpyDeviceToHost));
        // copy variance
        gpuErrchk(hipMemcpy((Scalar2 *)h_in + (unsigned long long)(lags.size() + 1) * _nx * ny + q_start,
                             d_var,
                             curr_chunk_size * sizeof(Scalar2),
                             hipMemcpyDeviceToHost));
    }

    // ***Free memory
    gpuErrchk(hipFree(d_workspace1));
    gpuErrchk(hipFree(d_workspace2));
    gpuErrchk(hipFree(d_lags));
    gpuErrchk(hipFree(d_power_spec));
    gpuErrchk(hipFree(d_var));
}

/*! \brief Convert to full and fftshifted image structure function on the GPU
    \param h_in             input array after structure function calculation
    \param Nlags            number of lags analyzed
    \param nx               number of fft nodes in x direction
    \param ny               number of fft nodes in y direction
    \param num_fullshift    number of full and shift chunks
    \param pitch_fs         pitch of device array for full and shift operations
 */
void make_full_shift(Scalar *h_in,
                     unsigned long long Nlags,
                     unsigned long long nx,
                     unsigned long long ny,
                     unsigned long long num_fullshift,
                     unsigned long long pitch_fs)
{
    int device_id;
    hipGetDevice(&device_id);

    unsigned long long _nx = nx / 2 + 1;                             // fft2 r2c number of complex elements over x
    unsigned long long chunk_size = (Nlags - 1) / num_fullshift + 1; // number of lags in a chunk

    // ***Allocate space on device
    // workspaces
    Scalar *d_workspace1, *d_workspace2;
    gpuErrchk(hipMalloc(&d_workspace1, pitch_fs * ny * chunk_size * 2 * sizeof(Scalar)));
    gpuErrchk(hipMalloc(&d_workspace2, pitch_fs * ny * chunk_size * 2 * sizeof(Scalar)));

    // ***Compute optimal kernels execution parameters
    dim3 blockSize_full(TILE_DIM, BLOCK_ROWS, 1);
    int maxGridSizeX, maxGridSizeY;
    gpuErrchk(hipDeviceGetAttribute(&maxGridSizeX, hipDeviceAttributeMaxGridDimX, device_id));
    gpuErrchk(hipDeviceGetAttribute(&maxGridSizeY, hipDeviceAttributeMaxGridDimY, device_id));
    int gridSize_full_x = min((_nx + TILE_DIM - 1) / TILE_DIM, (unsigned long long)maxGridSizeX);
    int gridSize_full_y = min((ny * chunk_size + TILE_DIM - 1) / TILE_DIM, (unsigned long long)maxGridSizeY);
    dim3 gridSize_full(gridSize_full_x, gridSize_full_y, 1);
    int gridSize_shift_x = min((nx + TILE_DIM - 1) / TILE_DIM, (unsigned long long)maxGridSizeX);
    int gridSize_shift_y = min((ny * chunk_size + TILE_DIM - 1) / TILE_DIM, (unsigned long long)maxGridSizeY);
    dim3 gridSize_shift(gridSize_shift_x, gridSize_shift_y, 1);

    // ***Loop over chunks
    for (unsigned long long chunk = 0; chunk < num_fullshift; chunk++)
    {
        // Get input offset
        unsigned long long ioffset = chunk * chunk_size * 2 * _nx * ny;
        // Get current chunk size
        unsigned long long curr_chunk_size = (chunk + 1) * chunk_size > Nlags ? Nlags - chunk * chunk_size : chunk_size;

        if (curr_chunk_size != chunk_size)
        {
            gridSize_full_y = min((ny * curr_chunk_size + TILE_DIM - 1) / TILE_DIM, (unsigned long long)maxGridSizeY);
            gridSize_shift_y = min((ny * curr_chunk_size + TILE_DIM - 1) / TILE_DIM, (unsigned long long)maxGridSizeY);
            gridSize_full.y = gridSize_full_y;
            gridSize_shift.y = gridSize_shift_y;
        }

        // ***Copy values from host to device
        gpuErrchk(hipMemcpy2D(d_workspace1,
                               pitch_fs * 2 * sizeof(Scalar),
                               h_in + ioffset,
                               2 * _nx * sizeof(Scalar),
                               2 * _nx * sizeof(Scalar),
                               curr_chunk_size * ny,
                               hipMemcpyHostToDevice));

        // ***Make full power spectrum (workspace1 --> workspace2)
        make_full_powspec_kernel<<<gridSize_full, blockSize_full>>>((Scalar2 *)d_workspace1,
                                                                    pitch_fs,
                                                                    d_workspace2,
                                                                    2 * pitch_fs,
                                                                    _nx,
                                                                    nx,
                                                                    ny,
                                                                    curr_chunk_size,
                                                                    (_nx + TILE_DIM - 1) / TILE_DIM,
                                                                    (ny * curr_chunk_size + TILE_DIM - 1) / TILE_DIM);
        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipDeviceSynchronize());

        // ***Shift power spectrum (workspace2 --> workspace1)
        shift_powspec_kernel<<<gridSize_shift, blockSize_full>>>(d_workspace2,
                                                                 2 * pitch_fs,
                                                                 d_workspace1,
                                                                 2 * pitch_fs,
                                                                 nx,
                                                                 ny,
                                                                 curr_chunk_size,
                                                                 (nx + TILE_DIM - 1) / TILE_DIM,
                                                                 (ny * curr_chunk_size + TILE_DIM - 1) / TILE_DIM);
        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipDeviceSynchronize());

        // ***Copy values from device to host (make contiguous on memory)
        // Get output offset
        unsigned long long ooffset = chunk * chunk_size * nx * ny;
        gpuErrchk(hipMemcpy2D(h_in + ooffset,
                               nx * sizeof(Scalar),
                               d_workspace1,
                               pitch_fs * 2 * sizeof(Scalar),
                               nx * sizeof(Scalar),
                               curr_chunk_size * ny,
                               hipMemcpyDeviceToHost));
    }

    // ***Free memory
    gpuErrchk(hipFree(d_workspace1));
    gpuErrchk(hipFree(d_workspace2));
}

/*!
    Compute image structure function using the WK theorem on the GPU
 */
void structure_function_fft(Scalar *h_in,
                            vector<unsigned int> lags,
                            unsigned long long length,
                            unsigned long long nx,
                            unsigned long long ny,
                            unsigned long long nt,
                            unsigned long long num_chunks,
                            unsigned long long pitch_q,
                            unsigned long long pitch_t,
                            unsigned long long pitch_nt)
{
    int device_id;
    hipGetDevice(&device_id);

    unsigned long long _nx = nx / 2 + 1;                             // fft2 r2c number of complex elements over x
    unsigned long long chunk_size = (_nx * ny - 1) / num_chunks + 1; // number of q points in a chunk

    // ***Allocate memory on device for fft
    Scalar *d_workspace1, *d_workspace2;
    // To speed up memory trasfer and meet the alignment requirements for coalescing,
    // we space each subarray by pitch1 (over q) or pitch2 (over t)
    gpuErrchk(hipMalloc(&d_workspace1, chunk_size * pitch_nt * 2 * sizeof(Scalar)));
    unsigned long long workspace2_size = max(chunk_size * pitch_t, pitch_q * length);
    gpuErrchk(hipMalloc(&d_workspace2, workspace2_size * 2 * sizeof(Scalar)));

    // helper arrays
    unsigned int *d_lags;
    gpuErrchk(hipMalloc(&d_lags, lags.size() * sizeof(unsigned int)));
    gpuErrchk(hipMemcpy(d_lags, lags.data(), lags.size() * sizeof(unsigned int), hipMemcpyHostToDevice));
    Scalar2 *d_power_spec, *d_var;
    gpuErrchk(hipMalloc(&d_power_spec, chunk_size * sizeof(Scalar2)));
    gpuErrchk(hipMalloc(&d_var, chunk_size * sizeof(Scalar2)));

    // ***Create fft plan
    hipfftHandle fft_plan = fft_create_plan(nt,
                                           chunk_size,
                                           pitch_nt);

    // ***Compute efficient kernels execution configurations
    int minGridSize; // The minimum grid size needed to achieve the
                     // maximum occupancy for a full device launch
    int numSMs;      // Number of streaming multiprocessors
    gpuErrchk(hipDeviceGetAttribute(&numSMs, hipDeviceAttributeMultiprocessorCount, device_id));

    // transpose_complex_matrix_kernel
    dim3 blockSize_tran(TILE_DIM, BLOCK_ROWS, 1);
    int maxGridSizeX, maxGridSizeY;
    gpuErrchk(hipDeviceGetAttribute(&maxGridSizeX, hipDeviceAttributeMaxGridDimX, device_id));
    gpuErrchk(hipDeviceGetAttribute(&maxGridSizeY, hipDeviceAttributeMaxGridDimY, device_id));
    int gridSize_tran1_x = min((chunk_size + TILE_DIM - 1) / TILE_DIM, (unsigned long long)maxGridSizeX);
    int gridSize_tran1_y = min((length + TILE_DIM - 1) / TILE_DIM, (unsigned long long)maxGridSizeY);
    dim3 gridSize_tran1(gridSize_tran1_x, gridSize_tran1_y, 1);
    int gridSize_tran2_x = min(((unsigned long long)(lags.size()) + TILE_DIM - 1) / TILE_DIM, (unsigned long long)maxGridSizeX);
    int gridSize_tran2_y = min((chunk_size + TILE_DIM - 1) / TILE_DIM, (unsigned long long)maxGridSizeY);
    dim3 gridSize_tran2(gridSize_tran2_x, gridSize_tran2_y, 1);

    // square_modulus_kernel
    int blockSize_sqmod; // The launch configurator returned block size
    int gridSize_sqmod;  // The actual grid size needed, based on input size
    gpuErrchk(hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize_sqmod, square_modulus_kernel, 0, 0));
    // Round up according to array size
    gridSize_sqmod = min((chunk_size + blockSize_sqmod - 1) / blockSize_sqmod, 32ULL * numSMs);

    // scale_array_kernel
    int blockSize_scale; // The launch configurator returned block size
    int gridSize_scale;  // The actual grid size needed, based on input size
    gpuErrchk(hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize_scale, scale_array_kernel, 0, 0));
    // Round up according to array size
    gridSize_scale = min((chunk_size + blockSize_scale - 1) / blockSize_scale, 32ULL * numSMs);

    // linear_combination_final_kernel
    int blockSize_final; // The launch configurator returned block size
    int gridSize_final;  // The actual grid size needed, based on input size
    gpuErrchk(hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize_final, linear_combination_final_kernel, 0, 0));
    // Round up according to array size
    gridSize_final = min(chunk_size, 32ULL * numSMs);

    // copy_selected_lags_kernel
    int blockSize_copy; // The launch configurator returned block size
    int gridSize_copy;  // The actual grid size needed, based on input size
    gpuErrchk(hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize_copy, copy_selected_lags_kernel, 0, 0));
    // Round up according to array size
    gridSize_copy = min(chunk_size, 32ULL * numSMs);

    // linear combination for power spectrum and variance
    int blockSize_lc; // The launch configurator returned block size
    int gridSize_lc;  // The actual grid size needed, based on input size
    gpuErrchk(hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize_lc, linear_combination_kernel, 0, 0));
    // Round up according to array size
    gridSize_lc = min((chunk_size + blockSize_lc - 1) / blockSize_lc, 32ULL * numSMs);

    // ***Loop over the chunks
    for (unsigned long long chunk = 0; chunk < num_chunks; chunk++)
    {
        // ***Get initial and final q indices
        unsigned long long q_start = chunk * chunk_size;
        unsigned long long q_end = (chunk + 1) * chunk_size < _nx * ny ? (chunk + 1) * chunk_size : _nx * ny;
        unsigned long long curr_chunk_size = q_end - q_start;

        // ***Check if fft plans are still ok (batch number must be the same as previous one)
        // This may be true during the very last iteration only
        if (curr_chunk_size != chunk_size)
        {
            cufftSafeCall(hipfftDestroy(fft_plan));
            fft_plan = fft_create_plan(nt,
                                       curr_chunk_size,
                                       pitch_nt);
            // also change gridSize for transpose and linear combination
            gridSize_tran1.x = (curr_chunk_size + TILE_DIM - 1) / TILE_DIM;
            gridSize_tran2.y = (curr_chunk_size + TILE_DIM - 1) / TILE_DIM;
            gridSize_final = min(curr_chunk_size, 32ULL * numSMs);
            gridSize_sqmod = min((curr_chunk_size + blockSize_sqmod - 1) / blockSize_sqmod, 32ULL * numSMs);
            gridSize_scale = min((curr_chunk_size + blockSize_scale - 1) / blockSize_scale, 32ULL * numSMs);
        }

        // ***Zero-out elements of workspace1 and workspace2 device arrays
        gpuErrchk(hipMemset2D(d_workspace1, 2 * pitch_nt * sizeof(Scalar), 0.0, 2 * nt * sizeof(Scalar), curr_chunk_size));
        gpuErrchk(hipMemset(d_workspace2, 0.0, 2 * workspace2_size * sizeof(Scalar)));

        // ***Copy values from host buffer to device workspace2 with pitch_q
        unsigned long long offset = 2 * q_start;                     // host source array offset
        unsigned long long spitch = 2 * (_nx * ny) * sizeof(Scalar); // host source array pitch
        unsigned long long dpitch = 2 * pitch_q * sizeof(Scalar);    // device destination array pitch
        gpuErrchk(hipMemcpy2D(d_workspace2,
                               dpitch,
                               h_in + offset,
                               spitch,
                               2 * curr_chunk_size * sizeof(Scalar),
                               length,
                               hipMemcpyHostToDevice));

        // ***Transpose complex matrix ({d_workspace2; pitch_q} --> {d_workspace1; pitch_nt})
        transpose_complex_matrix_kernel<<<gridSize_tran1, blockSize_tran>>>((Scalar2 *)d_workspace2,
                                                                            pitch_q,
                                                                            (Scalar2 *)d_workspace1,
                                                                            pitch_nt,
                                                                            curr_chunk_size,
                                                                            length,
                                                                            (curr_chunk_size + TILE_DIM - 1) / TILE_DIM,
                                                                            (length + TILE_DIM - 1) / TILE_DIM);
        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipDeviceSynchronize());

        // ***Copy values ({d_workspace1; pitch_nt} --> {d_workspace2; pitch_t})
        dpitch = 2 * pitch_t * sizeof(Scalar);
        spitch = 2 * pitch_nt * sizeof(Scalar);
        gpuErrchk(hipMemcpy2D(d_workspace2,
                               dpitch,
                               d_workspace1,
                               spitch,
                               length * 2 * sizeof(Scalar),
                               curr_chunk_size,
                               hipMemcpyDeviceToDevice));

        // +++ FFT PART +++
        // ***Do fft (d_workspace1 --> d_workspace1)
        cufftSafeCall(CufftExecC2C(fft_plan, (CUFFTCOMPLEX *)d_workspace1, (CUFFTCOMPLEX *)d_workspace1, HIPFFT_FORWARD));
        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipDeviceSynchronize());

        // ***Compute square modulus (d_workspace1 --> d_workspace1)
        square_modulus_kernel<<<gridSize_sqmod, blockSize_sqmod>>>((Scalar2 *)d_workspace1,
                                                                   nt,
                                                                   pitch_nt,
                                                                   curr_chunk_size);
        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipDeviceSynchronize());

        // ***Copy square modulus of sum part
        gpuErrchk(hipMemcpy2D(d_var,
                               sizeof(Scalar2),
                               d_workspace1,
                               pitch_nt * sizeof(Scalar2),
                               sizeof(Scalar2),
                               curr_chunk_size,
                               hipMemcpyDeviceToDevice));

        // ***Do fft (d_workspace1 --> d_workspace1)
        cufftSafeCall(CufftExecC2C(fft_plan, (CUFFTCOMPLEX *)d_workspace1, (CUFFTCOMPLEX *)d_workspace1, HIPFFT_FORWARD));
        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipDeviceSynchronize());

        // ***Scale fft part (d_workspace1 --> d_workspace1)
        scale_array_kernel<<<gridSize_scale, blockSize_scale>>>((Scalar2 *)d_workspace1,
                                                                pitch_nt,
                                                                nt,
                                                                1.0 / (double)nt,
                                                                curr_chunk_size);
        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipDeviceSynchronize());

        // ***Copy power spectrum part
        gpuErrchk(hipMemcpy2D(d_power_spec,
                               sizeof(Scalar2),
                               d_workspace1,
                               pitch_nt * sizeof(Scalar2),
                               sizeof(Scalar2),
                               curr_chunk_size,
                               hipMemcpyDeviceToDevice));

        // scale power spectrum (d_power_spec = d_power_spec * 1/length + 0)
        linear_combination_kernel<<<gridSize_lc, blockSize_lc>>>(d_power_spec,
                                                                 d_power_spec,
                                                                 make_double2(1.0 / (double)length, 0.0),
                                                                 d_power_spec,
                                                                 make_double2(0.0, 0.0),
                                                                 curr_chunk_size);

        // ***Compute variance (d_var = (1.0, 0.0) * d_power_spec + (-1.0/(length * length), 0.0) * d_var)
        linear_combination_kernel<<<gridSize_lc, blockSize_lc>>>(d_var,
                                                                 d_power_spec,
                                                                 make_double2(1.0, 0.0),
                                                                 d_var,
                                                                 make_double2(-1.0 / (double)(length * length), 0.0),
                                                                 curr_chunk_size);

        // +++ CUMULATIVE SUM PART +++
        // ***Compute square modulus (d_workspace2 --> d_workspace2)
        square_modulus_kernel<<<gridSize_sqmod, blockSize_sqmod>>>((Scalar2 *)d_workspace2,
                                                                   length,
                                                                   pitch_t,
                                                                   curr_chunk_size);

        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipDeviceSynchronize());

        // ***Copy the value into the imaginary part of the opposite (with respect to time) element (d_workspace2 --> d_workspace2)
        real2imagopposite_kernel<<<gridSize_sqmod, blockSize_sqmod>>>((Scalar2 *)d_workspace2,
                                                                      length,
                                                                      pitch_t,
                                                                      curr_chunk_size);
        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipDeviceSynchronize());

        // ***Compute (exclusive) cumulative sum (prefix scan)
        scan_wrap(d_workspace2,
                  d_workspace2,
                  2 * length,
                  2 * pitch_t,
                  curr_chunk_size);

        //  ***Linearly combine the two parts (workspace1 + workspace2 --> workspace2)
        linear_combination_final_kernel<<<gridSize_final, blockSize_final>>>((Scalar2 *)d_workspace2,
                                                                             pitch_t,
                                                                             (Scalar2 *)d_workspace1,
                                                                             pitch_nt,
                                                                             (Scalar2 *)d_workspace2,
                                                                             pitch_t,
                                                                             length,
                                                                             curr_chunk_size);
        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipDeviceSynchronize());

        // ***Keep only selected lags ({d_workspace2; pitch_t} --> {d_workspace1; pitch_t})
        copy_selected_lags_kernel<<<gridSize_copy, blockSize_copy>>>((Scalar2 *)d_workspace2,
                                                                     (Scalar2 *)d_workspace1,
                                                                     d_lags,
                                                                     lags.size(),
                                                                     pitch_t,
                                                                     pitch_t,
                                                                     curr_chunk_size);
        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipDeviceSynchronize());

        // ***Transpose complex matrix ({d_workspace1; pitch_t} --> {d_workspace2; pitch_q})
        transpose_complex_matrix_kernel<<<gridSize_tran2, blockSize_tran>>>((Scalar2 *)d_workspace1,
                                                                            pitch_t,
                                                                            (Scalar2 *)d_workspace2,
                                                                            pitch_q,
                                                                            lags.size(),
                                                                            curr_chunk_size,
                                                                            (lags.size() + TILE_DIM - 1) / TILE_DIM,
                                                                            (curr_chunk_size + TILE_DIM - 1) / TILE_DIM);
        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipDeviceSynchronize());

        // ***Copy from device to host (d_workspace2 --> host)
        // elements are treated as complex
        // to speed up transfer, use pitch_q
        offset = 2 * q_start;                     // host destination array offset
        spitch = 2 * pitch_q * sizeof(Scalar);    // host source array pitch
        dpitch = 2 * (_nx * ny) * sizeof(Scalar); // device destination array pitch
        gpuErrchk(hipMemcpy2D(h_in + offset,
                               dpitch,
                               d_workspace2,
                               spitch,
                               2 * curr_chunk_size * sizeof(Scalar),
                               lags.size(),
                               hipMemcpyDeviceToHost));

        // copy power spectrum
        gpuErrchk(hipMemcpy((Scalar2 *)h_in + (unsigned long long)(lags.size()) * _nx * ny + q_start,
                             d_power_spec,
                             curr_chunk_size * sizeof(Scalar2),
                             hipMemcpyDeviceToHost));
        // copy variance
        gpuErrchk(hipMemcpy((Scalar2 *)h_in + (unsigned long long)(lags.size() + 1) * _nx * ny + q_start,
                             d_var,
                             curr_chunk_size * sizeof(Scalar2),
                             hipMemcpyDeviceToHost));
    }

    // ***Free memory
    cufftSafeCall(hipfftDestroy(fft_plan));
    gpuErrchk(hipFree(d_workspace1));
    gpuErrchk(hipFree(d_workspace2));
    gpuErrchk(hipFree(d_lags));
    gpuErrchk(hipFree(d_power_spec));
    gpuErrchk(hipFree(d_var));
}
