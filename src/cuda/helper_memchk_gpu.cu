#include "hip/hip_runtime.h"
// Maintainer: enrico-lattuada

/*! \file helper_memchk_gpu.cu
    \brief Definition of CUDA helper functions for memory check and optimization for GPU routines
*/

// *** headers ***
#include "helper_memchk_gpu.cuh"
#include "helper_cufft.cuh"
#include "helper_debug.cuh"

#include <hip/hip_runtime.h>

// *** code ***

/*!
    Evaluate the device memory pitch for multiple subarrays of size N with 8bytes elements
*/
void cudaGetDevicePitch16B(size_t N,
                           size_t &pitch)
{
    double2 *d_arr;

    gpuErrchk(hipMallocPitch(&d_arr, &pitch, N * sizeof(double2), 2));

    pitch /= sizeof(double2);

    gpuErrchk(hipFree(d_arr));
}

/*!
    Evaluate the device memory pitch for multiple subarrays of size N with 8bytes elements
*/
void cudaGetDevicePitch8B(size_t N,
                          size_t &pitch)
{
    double *d_arr;

    gpuErrchk(hipMallocPitch(&d_arr, &pitch, N * sizeof(double), 2));

    pitch /= sizeof(double);

    gpuErrchk(hipFree(d_arr));
}

/*!
    Evaluate the device memory pitch for multiple subarrays of size N with 4bytes elements
*/
void cudaGetDevicePitch4B(size_t N,
                          size_t &pitch)
{
    float *d_arr;

    gpuErrchk(hipMallocPitch(&d_arr, &pitch, N * sizeof(float), 2));

    pitch /= sizeof(float);

    gpuErrchk(hipFree(d_arr));
}

/*!
    Evaluate the device memory pitch for multiple subarrays of size N with 2bytes elements
*/
void cudaGetDevicePitch2B(size_t N,
                          size_t &pitch)
{
    int16_t *d_arr;

    gpuErrchk(hipMallocPitch(&d_arr, &pitch, N * sizeof(int16_t), 2));

    pitch /= sizeof(int16_t);

    gpuErrchk(hipFree(d_arr));
}

/*!
    Evaluate the device memory pitch for multiple subarrays of size N with 2bytes elements
*/
void cudaGetDevicePitch1B(size_t N,
                          size_t &pitch)
{
    int8_t *d_arr;

    gpuErrchk(hipMallocPitch(&d_arr, &pitch, N * sizeof(int8_t), 2));

    pitch /= sizeof(int8_t);

    gpuErrchk(hipFree(d_arr));
}

/*!
    Evaluate the device memory size in bytes for fft2
*/
void cudaGetFft2MemSize(size_t nx,
                        size_t ny,
                        size_t batch,
                        size_t pitch,
                        bool is_double_prec,
                        size_t *memsize,
                        hipfftResult &cufft_res)
{
    fft2_get_mem_size(nx,
                      ny,
                      batch,
                      pitch,
                      is_double_prec,
                      memsize,
                      cufft_res);
}

/*!
    Evaluate the device memory size in bytes for fft
*/
void cudaGetFftMemSize(size_t nt,
                       size_t batch,
                       size_t pitch,
                       bool is_double_prec,
                       size_t *memsize,
                       hipfftResult &cufft_res)
{
    fft_get_mem_size(nt,
                     batch,
                     pitch,
                     is_double_prec,
                     memsize,
                     cufft_res);
}