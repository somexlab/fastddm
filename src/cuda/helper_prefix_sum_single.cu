#include "hip/hip_runtime.h"
// Maintainer: enrico-lattuada

/*! \file helper_prefix_sum_single.cu
    \brief Definition of helper functions for cumulative sum (prefix sum) on GPU (single precision)
*/

// *** headers ***
#include "helper_prefix_sum.cuh"
#include "helper_prefix_sum_single.cuh"
#include "helper_debug.cuh"

#include <hip/hip_runtime.h>

// definitions
const unsigned long long NUM_BANKS = 32;
const unsigned long long LOG_NUM_BANKS = 5;
//#ifdef ZERO_BANK_CONFLICTS
//#define CONFLICT_FREE_OFFSET(n) \ 
// ((n) >> NUM_BANKS + (n) >> (2 * LOG_NUM_BANKS))
//#else
#define CONFLICT_FREE_OFFSET(n) ((n) >> LOG_NUM_BANKS)
//#endif

const unsigned long long BLOCK_SIZE = 512;
const unsigned long long ELEMENTS_PER_BLOCK = BLOCK_SIZE * 2;

// *** code ***

// See:
// https://developer.download.nvidia.com/compute/cuda/1.1-Beta/x86_website/projects/scan/doc/scan.pdf
// https://github.com/lxxue/prefix_sum/blob/master/prefix_sum.cu

/*!
    Scan multiple large arrays on the GPU
 */
void scanManyLargeArrays(float *output,
                         float *input,
                         unsigned long long length,
                         unsigned long long dist,
                         unsigned long long N)
{
    unsigned long long Nx = length / ELEMENTS_PER_BLOCK;             // number of even blocks per row
    unsigned long long remainder = length - Nx * ELEMENTS_PER_BLOCK; // remainder from even blocks

    if (remainder == 0)
    {
        scanManyLargeEvenArrays(output,
                                input,
                                length,
                                dist,
                                Nx,
                                N);
    }
    else
    {
        unsigned long long length_even = length - remainder;

        // copy the last element of the even part of the subarray before prefix sum
        float *a1;
        gpuErrchk(hipMalloc(&a1, N * sizeof(float)));
        int numSMs;
        gpuErrchk(hipDeviceGetAttribute(&numSMs, hipDeviceAttributeMultiprocessorCount, 0));
        int gridSize_copy = min(N, 32ULL * numSMs);
        copy_every_kernel<<<gridSize_copy, 1>>>(a1,
                                                input + length_even - 1,
                                                dist,
                                                N);
        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipDeviceSynchronize());

        // scan over even arrays
        scanManyLargeEvenArrays(output,
                                input,
                                length_even,
                                dist,
                                Nx,
                                N);

        // scan the remaining elements
        scanManySmallArrays(output + length_even,
                            input + length_even,
                            remainder,
                            dist,
                            N);

        // copy the last element of the even part of the subarray after prefix sum
        float *a2;
        gpuErrchk(hipMalloc(&a2, N * sizeof(float)));
        copy_every_kernel<<<gridSize_copy, 1>>>(a2,
                                                output + length_even - 1,
                                                dist,
                                                N);
        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipDeviceSynchronize());

        // sum the last element of the even arrays portion
        // and the first element of the remainder input
        // to the remainder scan
        add_many_kernel<<<gridSize_copy, remainder>>>(output+length_even,
                                                      dist,
                                                      1,
                                                      N,
                                                      remainder,
                                                      a1,
                                                      a2);
        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipDeviceSynchronize());

        // free memory
        gpuErrchk(hipFree(a1));
        gpuErrchk(hipFree(a2));
    }
}

/*!
    Scan multiple large even arrays on the GPU
 */
void scanManyLargeEvenArrays(float *output,
                             float *input,
                             unsigned long long length,
                             unsigned long long dist,
                             unsigned long long Nx,
                             unsigned long long N)
{
    // compute execution parameters
    const unsigned long long shared_mem_size = ELEMENTS_PER_BLOCK * sizeof(float);
    int numSMs;
    gpuErrchk(hipDeviceGetAttribute(&numSMs, hipDeviceAttributeMultiprocessorCount, 0));
    int gridSize = min(Nx * N, 32ULL * numSMs);

    // allocate partial sums and incr arrays
    float *sums, *incr;
    gpuErrchk(hipMalloc(&sums, Nx * N * sizeof(float)));
    gpuErrchk(hipMalloc(&incr, Nx * N * sizeof(float)));

    // do scan
    prescan_many_even_kernel<<<gridSize, BLOCK_SIZE, 2 * shared_mem_size>>>(output,
                                                                            input,
                                                                            dist,
                                                                            Nx,
                                                                            Nx * N,
                                                                            ELEMENTS_PER_BLOCK,
                                                                            sums);
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());

    // compute increment
    unsigned long long scan_sum_threads_needed = (Nx + 1) / 2;
    if (scan_sum_threads_needed > BLOCK_SIZE)
    {
        scanManyLargeArrays(incr,
                            sums,
                            Nx,
                            Nx,
                            N);
    }
    else
    {
        scanManySmallArrays(incr,
                            sums,
                            Nx,
                            Nx,
                            N);
    }

    // add increment
    add_many_kernel<<<gridSize, ELEMENTS_PER_BLOCK>>>(output,
                                                      dist,
                                                      Nx,
                                                      Nx * N,
                                                      ELEMENTS_PER_BLOCK,
                                                      incr);
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());

    // free memory
    gpuErrchk(hipFree(sums));
    gpuErrchk(hipFree(incr));
}

/*!
    Scan multiple small arrays on the GPU
 */
void scanManySmallArrays(float *output,
                         float *input,
                         unsigned long long length,
                         unsigned long long dist,
                         unsigned long long N)
{
    unsigned long long powerOfTwo = nextPowerOfTwo(length);

    // Compute execution parameters
    const unsigned long long shared_mem_size = powerOfTwo * sizeof(float);
    int numSMs;
    gpuErrchk(hipDeviceGetAttribute(&numSMs, hipDeviceAttributeMultiprocessorCount, 0));
    int gridSize = min(N, 32ULL * numSMs);
    int blockSize = (length + 1) / 2;

    // do scan
    prescan_many_arbitrary_kernel<<<gridSize, blockSize, 2 * shared_mem_size>>>(output,
                                                                                input,
                                                                                dist,
                                                                                N,
                                                                                length,
                                                                                powerOfTwo);
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());
}

/*!
    Wrapper function to compute cumulative sum on multiple subarrays on the GPU
 */
void scan_wrap(float *output,
               float *input,
               unsigned long long length,
               unsigned long long dist,
               unsigned long long N)
{
    if (length > ELEMENTS_PER_BLOCK)
    {
        scanManyLargeArrays(output,
                            input,
                            length,
                            dist,
                            N);
    }
    else
    {
        scanManySmallArrays(output,
                            input,
                            length,
                            dist,
                            N);
    }
}

/*!
    Step 1 of cumsummany:
    Compute cumulative sum on multiple even portions of the subarrays on the GPU
 */
__global__ void prescan_many_even_kernel(float *output,
                                         float *input,
                                         unsigned long long dist,
                                         unsigned long long Nx,
                                         unsigned long long NxNy,
                                         unsigned long long n,
                                         float *sums)
{
    extern __shared__ float temp[];

    for (unsigned long long blockID = blockIdx.x; blockID < NxNy; blockID += gridDim.x)
    {
        unsigned long long threadID = threadIdx.x;
        unsigned long long y = (blockID / Nx);
        unsigned long long blockOffset = y * dist + (blockID - y * Nx) * n;

        // load input into shared memory
        unsigned long long ai = threadID;
        unsigned long long bi = threadID + (n / 2);
        unsigned long long bankOffsetA = CONFLICT_FREE_OFFSET(ai);
        unsigned long long bankOffsetB = CONFLICT_FREE_OFFSET(bi);
        temp[ai + bankOffsetA] = input[blockOffset + ai];
        temp[bi + bankOffsetB] = input[blockOffset + bi];

        // build sum in place up the tree
        unsigned long long offset = 1;
        for (unsigned long long d = n >> 1; d > 0; d >>= 1)
        {
            __syncthreads();
            if (threadID < d)
            {
                unsigned long long ai = offset * (2 * threadID + 1) - 1;
                unsigned long long bi = offset * (2 * threadID + 2) - 1;
                ai += CONFLICT_FREE_OFFSET(ai);
                bi += CONFLICT_FREE_OFFSET(bi);

                temp[bi] += temp[ai];
            }
            offset *= 2;
        }
        __syncthreads();

        // write total sum to sums and clear the last element
        if (threadID == 0)
        {
            sums[blockID] = temp[n - 1 + CONFLICT_FREE_OFFSET(n - 1)];
            temp[n - 1 + CONFLICT_FREE_OFFSET(n - 1)] = 0.0;
        }

        // traverse down tree and build scan
        for (unsigned long long d = 1; d < n; d *= 2)
        {
            offset >>= 1;
            __syncthreads();
            if (threadID < d)
            {
                unsigned long long ai = offset * (2 * threadID + 1) - 1;
                unsigned long long bi = offset * (2 * threadID + 2) - 1;
                ai += CONFLICT_FREE_OFFSET(ai);
                bi += CONFLICT_FREE_OFFSET(bi);

                float t = temp[ai];
                temp[ai] = temp[bi];
                temp[bi] += t;
            }
        }
        __syncthreads();

        output[blockOffset + ai] = temp[ai + bankOffsetA];
        output[blockOffset + bi] = temp[bi + bankOffsetB];
    }
}

/*!
    Step 2 of cumsummany:
    Compute cumulative sum on multiple arbitrary (small) portions of the subarrays on the GPU
 */
__global__ void prescan_many_arbitrary_kernel(float *output,
                                              float *input,
                                              unsigned long long dist,
                                              unsigned long long N,
                                              unsigned long long n,
                                              unsigned long long powerOfTwo)
{
    extern __shared__ float temp[];

    for (unsigned long long blockID = blockIdx.x; blockID < N; blockID += gridDim.x)
    {
        unsigned long long threadID = threadIdx.x;
        unsigned long long blockOffset = blockID * dist;

        // load input into shared memory
        unsigned long long ai = threadID;
        unsigned long long bi = threadID + (n / 2);
        unsigned long long bankOffsetA = CONFLICT_FREE_OFFSET(ai);
        unsigned long long bankOffsetB = CONFLICT_FREE_OFFSET(bi);

        if (threadID < n)
        {
            temp[ai + bankOffsetA] = input[blockOffset + ai];
            temp[bi + bankOffsetB] = input[blockOffset + bi];
        }
        else
        {
            temp[ai + bankOffsetA] = 0.0;
            temp[bi + bankOffsetB] = 0.0;
        }

        // build sum in place up the tree
        unsigned long long offset = 1;
        for (unsigned long long d = powerOfTwo >> 1; d > 0; d >>= 1)
        {
            __syncthreads();
            if (threadID < d)
            {
                unsigned long long ai = offset * (2 * threadID + 1) - 1;
                unsigned long long bi = offset * (2 * threadID + 2) - 1;
                ai += CONFLICT_FREE_OFFSET(ai);
                bi += CONFLICT_FREE_OFFSET(bi);

                temp[bi] += temp[ai];
            }
            offset *= 2;
        }
        __syncthreads();

        // clear the last element
        if (threadID == 0)
        {
            temp[powerOfTwo - 1 + CONFLICT_FREE_OFFSET(powerOfTwo - 1)] = 0;
        }

        // traverse down tree and build scan
        for (unsigned long long d = 1; d < powerOfTwo; d *= 2)
        {
            offset >>= 1;
            __syncthreads();
            if (threadID < d)
            {
                unsigned long long ai = offset * (2 * threadID + 1) - 1;
                unsigned long long bi = offset * (2 * threadID + 2) - 1;
                ai += CONFLICT_FREE_OFFSET(ai);
                bi += CONFLICT_FREE_OFFSET(bi);

                float t = temp[ai];
                temp[ai] = temp[bi];
                temp[bi] += t;
            }
        }
        __syncthreads();

        if (threadID < n)
        {
            output[blockOffset + ai] = temp[ai + bankOffsetA];
            output[blockOffset + bi] = temp[bi + bankOffsetB];
        }
    }
}

__global__ void add_many_kernel(float *output,
                                unsigned long long dist,
                                unsigned long long Nx,
                                unsigned long long NxNy,
                                unsigned long long n,
                                float *a)
{
    for (unsigned long long blockID = blockIdx.x; blockID < NxNy; blockID += gridDim.x)
    {
        unsigned long long threadID = threadIdx.x;
        if (threadID < n)
        {
            unsigned long long y = (blockID / Nx);
            unsigned long long blockOffset = y * dist + (blockID - y * Nx) * n;

            output[blockOffset + threadID] += a[blockID];
        }
    }
}

__global__ void add_many_kernel(float *output,
                                unsigned long long dist,
                                unsigned long long Nx,
                                unsigned long long NxNy,
                                unsigned long long n,
                                float *a1,
                                float *a2)
{
    for (unsigned long long blockID = blockIdx.x; blockID < NxNy; blockID += gridDim.x)
    {
        unsigned long long threadID = threadIdx.x;
        if (threadID < n)
        {
            unsigned long long y = (blockID / Nx);
            unsigned long long blockOffset = y * dist + (blockID - y * Nx) * n;

            output[blockOffset + threadID] += a1[blockID] + a2[blockID];
        }
    }
}

__global__ void copy_every_kernel(float *output,
                                  float *input,
                                  unsigned long long dist,
                                  unsigned long long N)
{
    for (unsigned long long blockID = blockIdx.x; blockID < N; blockID += gridDim.x)
    {
        output[blockID] = input[blockID * dist];
    }
}
